#include "hip/hip_runtime.h"

#include<chrono>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<float.h>
#include<time.h>
#include<iostream>
#include<vector>
#include<boost/math/special_functions/digamma.hpp>

#include<hip/hip_runtime.h>
#include<hipblas.h>

#ifdef PROF
#include<hip/hip_runtime_api.h>
#endif

#include"setting.hpp"
#include"cu_sum.hpp"
#include"profile.hpp"
#include"print.hpp"
#include"IO.hpp"

#define B0  1                 /* 以下はBernoulli数 */
#define B1  (-1.0 / 2.0)
#define B2  ( 1.0 / 6.0)
#define B4  (-1.0 / 30.0)
#define B6  ( 1.0 / 42.0)
#define B8  (-1.0 / 30.0)
#define B10 ( 5.0 / 66.0)
#define B12 (-691.0 / 2730.0)
#define B14 ( 7.0 / 6.0)
#define B16 (-3617.0 / 510.0)

template<typename type>
__global__ void divide_arr( type *arr , unsigned int size , ITR_SIZE div_val  ){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	arr[idx] = arr[idx] / div_val;
}

template<typename type >
__global__
void set_value_matrix( type *mat , ITR_SIZE rows  , ITR_SIZE cols , const type val ){
	const ITR_SIZE row = blockIdx.x * blockDim.x + threadIdx.x;
	for( ITR_SIZE col = 0 ; col < cols ; col++ ){
		mat[ col*rows + row] = val;
	}
}

__host__ __device__ float digamma(float xx ){
	float v, w;
	v = 0;
	while(xx < 100){
		v += 1/xx;
		xx++;
	}
	w = 1/(xx * xx);
	v += ((((((((B16 / 16) * w + (B14 /14)) * w + (B12 / 12)) * w + (B10 / 10)) * w + (B8 / 8)) * w + (B6 / 6)) * w + (B4 / 4))* w + (B2 / 2)) * w + 0.5 / xx;
	return log(xx) - v;
}

template<typename type>
__global__ void pow_self_elements( type *mat , const int rows , const int cols ){
	const int row = blockIdx.x * blockDim.x + threadIdx.x;

	if( row < rows )
		for( int col = 0 ; col < cols ; col++ ) mat[ row * cols + col ] = powf(mat[row * cols + col ] , 2);
}

template<typename type >
__global__ void compute_log_gaussian_prob( type *dev_r_ , type *dev_XK_, type *dev_K_ , type *dev_log_det_chol_ , type *dev_degree_of_freedom_ , const ITR_SIZE N , const int K , const int D ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ type share[];

	if( n < N ){

		if( threadIdx.x < K ){
			share[threadIdx.x] = dev_log_det_chol_[threadIdx.x] -  0.5 * D * logf(dev_degree_of_freedom_[threadIdx.x]);
		}

		__syncthreads();

		for( int k = 0 ; k < K ; k++ ){
			//dev_r_[k*N+n] = -0.5 * ( D * logf( 2 * M_PI ) + ( dev_r_[k*N+n] - ( 2 * dev_XK_[k*N+n]) + dev_K_[k] ) )  + dev_log_det_chol_[k] -  0.5 * D * logf(dev_degree_of_freedom_[k]);
			dev_r_[k*N+n] = -0.5 * ( D * logf( 2 * M_PI ) + ( dev_r_[k*N+n] - ( 2 * dev_XK_[k*N+n]) + dev_K_[k] ) ) + share[k];
		}
	}
}

__global__ void get_sum_of_2dim( float *devK , float *dev_KD_ , int K , int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;

	if( k < K ){
		devK[k] = 0;
		for( int d = 0 ; d < D ; d++ ){
			devK[k] += dev_KD_[ d * K + k ];
		}
	}

}

template<typename type>
__global__ void compute_weighted_log_prob( type *dev_log_gauss_, type *dev_log_lambda_ , type *dev_log_weights_ , const ITR_SIZE N , const int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ double sum_lw[];

	if( n < N ){

		if( threadIdx.x < K )
			sum_lw[threadIdx.x] = (double)dev_log_lambda_[threadIdx.x] + dev_log_weights_[threadIdx.x];
		__syncthreads();

		for( int k = 0 ; k < K ; k++ )
			//dev_log_gauss_[ k * N + n ] = dev_log_gauss_[ k * N + n ] + dev_log_lambda_[k] + dev_log_weights_[k];
			dev_log_gauss_[ k * N + n ] = dev_log_gauss_[ k * N + n ] + sum_lw[k];
	}

}

template<unsigned const int threadNum>
__global__ void compute_log_resp( float *norm , float *resp , ITR_SIZE N , int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	if( n < N ){

		__shared__ float _max_of_kth[threadNum];

		_max_of_kth[threadIdx.x] = resp[n];
		for( int k = 1 ; k < K ; k++ ) _max_of_kth[threadIdx.x] = max( _max_of_kth[threadIdx.x] , resp[ k * N + n ] );
		float _sum = 0.0;
#ifdef FMATHA
		for( int k = 0 ; k < K ; k++ ) _sum += expf( __fsub_rd ( resp[k*N+n] , _max_of_kth[threadIdx.x] ));
		const float _logsumexp = logf( _sum ) + _max_of_kth[threadIdx.x];
		for( int k = 0 ; k < K ; k++ ) resp[k*N+n] = __fsub_rd ( resp[k*N+n] , _logsumexp );
#else
		for( int k = 0 ; k < K ; k++ ) _sum += expf( resp[k*N+n] - _max_of_kth[threadIdx.x] );
		const float _logsumexp = logf( _sum ) + _max_of_kth[threadIdx.x];
		for( int k = 0 ; k < K ; k++ ) resp[k*N+n] =  resp[k*N+n] - _logsumexp ;
#endif
	}
}

template<typename type>
__global__ void pow_substitution_elements( type *to_mat , type *from_mat , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			to_mat[ d * K + k ] = powf( from_mat[d*K+k] , 2 );
		}
	}
}

template<typename type>
__global__ void hadamard_product_self( type *mat_a , type *mat_b , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			mat_a[d*K+k] = mat_a[d*K+k] * mat_b[d*K+k];
		}
	}
}

template<typename type>
__global__ void hadamard_product_subtitution( type *to_mat , type *from_a , type *from_b, const int  K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			to_mat[d * K + k] = from_a[d*K+k] * from_b[d*K+k];
		}
	}
}

//calc log Lambda for calc log resp
template<typename type>
__global__ void estimate_log_lambda_( float *log_lambda , float *degree_of_freedom_,  float *mean_precision_, const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ type share[];
	if ( k < K  ){
		double sum_digamma_nu_ = 0.0;

		if(threadIdx.x < K )
			share[k] = degree_of_freedom_[threadIdx.x] + 1;

		for( unsigned int d = 0 ; d < D ; d++ ){
			//digamma fuction
			sum_digamma_nu_ += logf(0.5*(share[k]-d) + 0.4849142) -  1 / ( 0.5*(share[k]-d) * 1.0271785 ) ;
		}
		log_lambda[k] = 0.5 * ( D * logf(2.0) + sum_digamma_nu_ - D / mean_precision_[k] ) ;
	}
}

template<const unsigned int threadNum , typename in_type , typename out_type>
__global__ void substitute_sum_with_idx( in_type *g_idata , out_type *g_odata , const ITR_SIZE N , const ITR_SIZE k){
	__shared__ out_type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	out_type mysum = (i < N) ? g_idata[i] : 0;
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x];
	sdata[tid] = mysum;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x + k] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}

template<typename in_type , typename out_type >
__global__ void cpy( out_type *to , in_type *from, const ITR_SIZE N , const ITR_SIZE k){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	to[idx] = from[k * N + idx];
}

//double ver.
template<const unsigned int threadNum , typename in_type , typename out_type >
__global__ void cpy_test( in_type *g_idata, out_type *g_odata, const ITR_SIZE N , const unsigned int k ){
	__shared__ out_type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	out_type mysum = (i < N) ? g_idata[ k * N + i ] : 0;
	if ( i +  blockDim.x < N) mysum += g_idata[ ( k * N + i ) +  blockDim.x  ]  ;
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}



template<typename in_type , typename out_type>
__global__ void calc_for_get_covariance_prior( out_type *dev_N_ , in_type *dev_X_, in_type *dev_mean_prior_ , const ITR_SIZE N , const ITR_SIZE d){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	if( idx < N )
		dev_N_[idx] = pow( dev_X_[d * N + idx] - dev_mean_prior_[d]  , 2 );
}

template<typename out_type>
__global__ void calc_for_get_covariance_prior( out_type *dev_N_ , float *dev_X_, float *dev_mean_prior_ , const ITR_SIZE N , const ITR_SIZE d ){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < N )
		dev_N_[idx] = powf( dev_X_[d * N + idx] - dev_mean_prior_[d]  , 2 );

}

//2 type of cpy_and_process_for_calc_sum_resp
template<typename type , const unsigned int threadNum>
__global__ void cpy_and_process_for_calc_sum_resp( type *from , float *to , const ITR_SIZE N  , const int k ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;


	if( n < N ){

#ifdef FMATHB
		to[n] = (float) (__expf( from[k * N + n ] ) * (double)from[ k * N + n ]);
#else 
		to[n] = (float) (expf( from[k * N + n ] ) * (double)from[ k * N + n ]);
#endif

#ifdef FMATHG //非推奨
		from[k*N+n] = __expf(from[k*N+n]);
#else
		from[k*N+n] = expf(from[k*N+n]);
#endif
#ifdef DEBUG
		CHECK_ERROR("cpy_and_process_for_calc_sum_resp" , from[k*N+n]);
#endif
	}

}
template<typename type , const unsigned int threadNum>
__global__ void cpy_and_process_for_calc_sum_resp( type *from , double *to , const ITR_SIZE N  , const int k ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;


	if( n < N ){
		__shared__ double sdata[threadNum];
		sdata[threadIdx.x] = exp( (double)from[k * N + n ] );
		to[n] = sdata[threadIdx.x] * from[ k * N + n ];
		from[k*N+n] = sdata[threadIdx.x];
#ifdef DEBUG
		CHECK_ERROR("cpy_and_process_for_calc_sum_resp" , from[k*N+n]);
#endif
	}
}

__global__ void divide_and_check_XBar( float *xk , float *nk,  float *mean_prior_ , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ float share[];

	if(threadIdx.x < K ) share[threadIdx.x] = nk[threadIdx.x];
	__syncthreads();


	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){

			if( share[k] > 1.0 )
				xk[d*K+k] = (double)xk[d*K+k] / share[k];
			else
				xk[d*K+k] = mean_prior_[k];

#ifdef DEBUG
			CHECK_ERROR("divide_and_check_XBar" , xk[d*K+k] );
#endif

		}
	}
}

__global__ void estimate_gaussian_covariance_diag( float *avg_X2 , float *avg_means_2 , float *avg_X_means , float *nk , float reg_covar , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	for( int d = 0 ; d < D ; d++ ){

		if( nk[k] > 0 )
			avg_X2[d*K+k] = ((double)avg_X2[d*K+k]/nk[k]) - 2 * ( (double)avg_X_means[d*K+k]/nk[k]) + avg_means_2[d*K+k];
		else
			avg_X2[d*K+k] = 1e-6;
		// prevent that S takeing negative value
		if( avg_X2[d*K+k] <= 0 ) {

#ifdef DEBUG
			printf("COLLECTION:estimate_gaussian_covariance_diag\n");
			CHECK_ERROR("estimate_gaussian_covariance_diag" , avg_X2[d*K+k]);
#endif
			avg_X2[d*K+k] = 1e-6;
		}
	}
}

__global__ void estimate_precisions( float *nk , float *xk, float *sk, float *mean_prior , float *precisions_cholesky , float *degree_of_freedom , float *covariance_prior , float *mean_precision ,  float *mean_precision_prior , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){


		if( nk[k] > 1.0 ){

			double diff[DIM] = {};
			for( int d = 0 ; d < D ; d++ ) diff[d] =  pow( xk[d*K+k] - mean_prior[d] , 2) ;

			for( int d = 0 ; d < D ; d++ ){
				diff[d] *= *mean_precision_prior / mean_precision[k];
				sk[d*K+k] += diff[d];
			}

			for( int d = 0 ; d < D ; d++ ){
				diff[d] = covariance_prior[d] + (double)nk[k] * sk[d*K+k];
			}

			//diff <- covariances
			for( int d = 0 ; d < D ; d++ ){
				diff[d] /= degree_of_freedom[k];
				precisions_cholesky[d*K+k] = 1 / sqrt(diff[d]);

#ifdef DEBUG
				CHECK_ERROR( "estimate_precisions" ,  precisions_cholesky[d*K+k]);
#endif
			}
		}
		else{
			for( int d = 0 ; d < D ; d++ )
				precisions_cholesky[d*K+k] = 1e-6;
		}
	}
}

template<const unsigned int threadNum , typename type>
__global__ void add_sum_resp( type *g_idata , double *sum_resp , const ITR_SIZE N){
	__shared__ type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	type mysum = 0;
	if( i < N ){
		mysum = g_idata[i];
#ifdef DEBUG
		CHECK_ERROR("add_sum_resp" , mysum );
#endif
	}
	if( i + blockDim.x < N ){
#ifdef FMATHC
		mysum = __fadd_rd ( mysum , g_idata[i + blockDim.x ] );
#else
		mysum = mysum , g_idata[i + blockDim.x ];
#endif
#ifdef DEBUG
		CHECK_ERROR("add_sum_resp" , mysum );
#endif
	}
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
#ifdef FMATHC
			sdata[tid] = mysum = __fadd_rd( mysum , sdata[tid + s] );
#else
			sdata[tid] = mysum = mysum + sdata[tid + s];
#endif
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
#ifdef FMATHD
			mysum = __fadd_rd( mysum , __shfl_down_sync(FULL_MASK, mysum , offset) );
#else
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
#endif
		}
	}
	//sum_respが大きくなっている場合が多いので倍精度で加算
	if (tid == 0) *sum_resp = *sum_resp + (double)mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}


/*** 2type of get_sum_of_N ***/
//float ver.
template<const unsigned int threadNum>
__global__ void get_sum_of_N( float *g_idata , float *g_odata , const ITR_SIZE N ){
	__shared__ float sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	float mysum = (i < N) ? g_idata[i] : 0;
#ifdef FMATHE
	if (i + blockDim.x < N) mysum = __fadd_rd ( mysum , g_idata[i + blockDim.x] );
#else
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x]  ;
#endif
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
#ifdef FMATHE
			sdata[tid] = mysum = __fadd_rd ( mysum , sdata[tid + s] );
#else
			sdata[tid] = mysum = mysum + sdata[tid + s];
#endif
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
#ifdef FMATHF
			mysum = __fadd_rd ( mysum , __shfl_down_sync(FULL_MASK, mysum , offset) );
#else
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
#endif
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}
//double ver.
template<const unsigned int threadNum>
__global__ void get_sum_of_N( double *g_idata , double *g_odata , const ITR_SIZE N ){
	__shared__ double sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	double mysum = (i < N) ? g_idata[i] : 0;
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x]  ;
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}

/*** end 2type of get_sum_of_N ***/

template<typename type>
__global__ void compute_log_det_chol( type *_log_det_chol , type *precision_chol, type *degree_of_freedom_ , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		double sum = 0.0;
		for( int d = 0 ; d < D ; d++ ) sum += log( double( precision_chol[d*K+k])  );
		// for( int d = 0 ; d < D ; d++ ) sum += logf( precision_chol[k] ); なぜかバグらない
		_log_det_chol[k] = sum;
	}

}

template<typename type>
__global__ void substitute_zero( type *a ){
	*a = 0.0;
}


template<typename in_type , typename out_type >
__global__ void get_sum_of_K( in_type *resp  , out_type *dev_N_ , ITR_SIZE N , int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	if( n < N ){
		float _max_of_kth = resp[n];
		for( int k = 1 ; k < K ; k++ ) _max_of_kth = max( _max_of_kth , resp[ k * N + n ] );
		double _sum = 0.0;
		for( int k = 0 ; k < K ; k++ ) _sum += exp( double(resp[k*N+n]) - _max_of_kth );
		dev_N_[n] = log(_sum) + _max_of_kth;
	}
}


template<typename type , typename n_type , unsigned int threadNum_N >
class vbgmm{

	public:
		size_t type_width;
		size_t size_means_ , size_covariances_ , size_precisions_ , size_precision_cholesky_ , size_weights_ , size_weight_concentration_ , size_mean_prior_ , size_degree_of_freedom_ , size_covariance_prior_ , size_mean_precision_ , size_gauss_ , size_X_ , size_lambda_ , size_r_ , size_log_det_chol ;

		//declare hst arrays
		type *means_ , *covariances_ , *precision_cholesky_ , *weights_ , *mean_prior_ , *degree_of_freedom_ ,  *mean_precision_ , *weight_concentration_prior_ , *mean_precision_prior_ , *degree_of_freedom_prior_  ;
		std::vector<type> weight_concentration_;
#ifdef INIT_MEAN_ON_HOST
		type *covariance_prior_;
#endif

		//declare dev arrays
		type	*dev_means_ , *dev_log_resp_ , *dev_X_ , *dev_log_weights_ , *dev_r_ , *dev_log_lambda_ , *dev_precisions_ , *dev_K_, *dev_XK_ , *dev_KD_ , *dev_KD_2_ ,  *dev_log_det_chol_  , *dev_degree_of_freedom_  , *dev_XD_ , *dev_mean_precision_  ,
					*dev_mean_prior_ ,  *dev_covariance_prior ,   *dev_mean_precision_prior;
		n_type *dev_N_;
		double *dev_1_;

		hipStream_t st_X_ , st_lambda_ , st_weights_ , st_log_lambda_ , st_log_weights_ , st_r_  , st_precisions_ , st_log_det_chol_ , st_KD_ , st_means_ , st_degree_of_freedom_ , st_mean_precision_  , st_prior_;

		bool converged = false;
		int n_iter , K , D , fin_iter = -1;
		ITR_SIZE N;

		//CUDA's parameter
		dim3 block_N = dim3( threadNum_N , 1 , 1) , grid_N;
		unsigned int threadNum_K;
		dim3 block_K , grid_K;
		double lower_bound , log_likelihood;

		const type _a = 1.0 , _b = 0.0;

		vbgmm( const type *input_X , const ITR_SIZE init_N , const unsigned int init_K , const unsigned int init_D , const unsigned int iter ){

			START_TIME(CONST);

			//std::cout << __FUNCTION__ << std::endl;
			N = init_N;
			K = init_K;
			D = init_D;

			if( N % block_N.x == 0 )
				grid_N = dim3( N / block_N.x  , 1 , 1 );
			else
				grid_N = dim3( N / block_N.x + 1 , 1 , 1 );

			threadNum_K = min ( K , 1024 );
			block_K = dim3(threadNum_K , 1 , 1 );
			grid_K = dim3( K / block_K.x  , 1 , 1);

			//set size of sum arrays
			type_width = sizeof(type);
			size_means_ = type_width * K * D;
			size_covariances_ = type_width * K * D;
			size_precisions_ = type_width * K * D;
			size_precision_cholesky_ = type_width * K * D;
			size_weights_ = type_width * K;
			size_weight_concentration_ = type_width * K;
			size_mean_prior_ = type_width * D;
			size_degree_of_freedom_ = type_width * K;
			size_covariance_prior_ = type_width * D;
			size_mean_precision_ = type_width * K;
			size_X_ = type_width * N * D;
			size_lambda_ = type_width * K;
			size_weights_ = type_width * K;
			size_r_ = type_width * N * K;
			size_log_det_chol = type_width * K;

			//malloc hst vec
			END_TIME(CONST);
			START_TIME(HOST_MALLOC);

			hipHostMalloc( &means_ , size_means_ );
			hipHostMalloc( &covariances_ , size_covariances_ );
			hipHostMalloc( &precision_cholesky_ , size_precision_cholesky_ );
			hipHostMalloc( &weights_ , size_weights_ );
			hipHostMalloc( &mean_prior_ , size_mean_prior_ );
			hipHostMalloc( &degree_of_freedom_ , size_degree_of_freedom_);
			hipHostMalloc( &mean_precision_ , size_mean_precision_);
			hipHostMalloc( &weight_concentration_prior_ , type_width );
			hipHostMalloc( &mean_precision_prior_ , type_width );
			hipHostMalloc( &degree_of_freedom_prior_ , type_width);
#ifdef INIT_MEAN_ON_HOST
			hipHostMalloc( &covariance_prior_ , size_covariance_prior_);
#endif

			END_TIME(HOST_MALLOC);
			START_TIME(DEVICE_MALLOC);

			//malloc & init weight_concentration_ as 1/K
			weight_concentration_.assign( K , (float)1/K );

			//malloc dev vec
			hipMalloc( &dev_X_ , size_X_ );
			hipMalloc( &dev_log_lambda_ , size_lambda_ );
			hipMalloc( &dev_means_ , size_means_ );
			hipMalloc( &dev_log_weights_ , size_weights_ );
			hipMalloc( &dev_r_ , size_r_ );
			hipMalloc( &dev_precisions_ , size_precisions_ );
			hipMalloc( &dev_K_ , type_width * K);
			hipMalloc( &dev_XK_ , type_width * N * K );
			hipMalloc( &dev_KD_ , type_width * K * D );
			hipMalloc( &dev_KD_2_ , type_width * K * D );
			hipMalloc( &dev_XD_ , type_width * N * D );
			hipMalloc( &dev_log_det_chol_ , size_log_det_chol );
			hipMalloc( &dev_degree_of_freedom_ , size_degree_of_freedom_ );
			hipMalloc( &dev_mean_precision_ , size_mean_precision_ );
			hipMalloc( &dev_mean_prior_ , size_mean_prior_ );  
			hipMalloc( &dev_covariance_prior , size_covariance_prior_ );
			hipMalloc( &dev_mean_precision_prior , type_width );
			hipMalloc( &dev_1_ , sizeof(double) );

			hipMalloc( &dev_N_ , sizeof(n_type)* N );

			hipStreamCreate( &st_X_ );
			hipStreamCreate( &st_lambda_);
			hipStreamCreate( &st_weights_);
			hipStreamCreate( &st_log_lambda_);
			hipStreamCreate( &st_log_weights_);
			hipStreamCreate( &st_r_ );
			hipStreamCreate( &st_precisions_);
			hipStreamCreate( &st_log_det_chol_);
			hipStreamCreate( &st_KD_ );
			hipStreamCreate( &st_means_ );
			hipStreamCreate( &st_degree_of_freedom_ );
			hipStreamCreate( &st_mean_precision_ );
			hipStreamCreate( &st_prior_ );

			END_TIME(DEVICE_MALLOC);
			START_TIME2(CONST);

			//tranport X
			hipMemcpyAsync( dev_X_ , input_X , size_X_ , hipMemcpyHostToDevice , st_X_ );

			//dev_XD_をX ** 2 で初期化
			pow_substitution_elements <<< grid_N , block_N , 0 , st_X_ >>> ( dev_XD_ , dev_X_ , N , D );

			//init weight_concentration_ as 1/K
			*weight_concentration_prior_ = (type)1/K; //alpha_0

			//init mean_precision_ as 1.0 & tranfer
			for( int k = 0 ; k < K ; k++ ) mean_precision_[k] = 1.0;
			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );

			//init mean_precision_prior_ as 1.0
			*mean_precision_prior_ = 1.0; // beta_0
			hipMemcpyAsync( dev_mean_precision_prior , mean_precision_prior_ , type_width , hipMemcpyHostToDevice , st_prior_ );

			//init degree_of_freedom_prior_ as D
			*degree_of_freedom_prior_ = D; //Nu_0

#ifndef INIT_MEAN_ON_HOST
			hipStreamSynchronize(st_X_);

			for( int d = 0 ; d < D ; d++ ){
				ITR_SIZE size = N;
				//cpy <type , n_type> <<< N / threadNum_N , threadNum_N >>> ( dev_N_ , dev_X_ , N , d);
				cpy_test <threadNum_N , type , n_type > <<< size / threadNum_N , threadNum_N >>> (  dev_X_ , dev_N_ ,N , d );
				size /= threadNum_N;
				hipDeviceSynchronize();

				if(size == 1 ){
					cpy <<<  1 , 1 >>> ( &dev_mean_prior_[d] , dev_N_ , 1 , 0  );
					continue;
				}

				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx<threadNum_N , n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_mean_prior_ , size , d);
					}
					else{
						get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			divide_arr<type> <<< D , ( D / threadNum_N) + 1 , 0 , st_means_ >>> ( dev_mean_prior_ , D ,  N );
			hipMemcpyAsync( mean_prior_ , dev_mean_prior_ , size_mean_prior_ , hipMemcpyDeviceToHost , st_prior_ );

			for( int d = 0 ; d < D ; d++ ){
				ITR_SIZE size = N;
				calc_for_get_covariance_prior <type , n_type > <<< N / threadNum_N , threadNum_N , 0 , st_means_>>> ( dev_N_ , dev_X_ , dev_mean_prior_ , N , d);
				hipDeviceSynchronize();
				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx<threadNum_N, n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_covariance_prior , size , d);
					}
					else{
						get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			divide_arr<type> <<< D , ( D / threadNum_N) + 1 >>> ( dev_covariance_prior , D ,  N );

#else
			//init mean_prior_ as X's mean
			for( int d = 0 ; d < D ; d++ ){
				double sum = 0.0;
				for( ITR_SIZE n = 0 ; n < N ; n++ ) sum += input_X[ d*N + n];
				mean_prior_[d] = (float)(sum/N);
			}
			hipMemcpyAsync( dev_mean_prior_ , mean_prior_ , size_mean_prior_ , hipMemcpyHostToDevice , st_prior_ );

			//init covariance_prior_ as X's cov
			for( int d = 0 ; d < D ; d++ ){
				double sum = 0.0;
				for( ITR_SIZE n = 0 ; n < N ; n++ ) sum += powf( input_X[d*N+n] - mean_prior_[d] , 2 );
				covariance_prior_[d] = (float)(sum/N);
			}
			hipMemcpyAsync( dev_covariance_prior , covariance_prior_ , size_covariance_prior_ , hipMemcpyHostToDevice , st_prior_ );
#endif

			//init degree_of_freedom_ as 1.0
			for( int k = 0 ; k < K ; k++ ) degree_of_freedom_[k] = 1.0; // Nu
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_);

			//set number of iteration
			n_iter = iter;

			END_TIME2(CONST);

		}


		//free arrays that were malloced
		void free_memory_d(){
			START_TIME(FREE);
			_free_dev_memory();
			_destroy_stream();
			END_TIME(FREE);
		}

		void free_memory_h(){
			_free_hst_memory();
		}

		void print_param(){
			int cnt = 0;
			bool f[K];
			for( int k = 0 ; k < K ; k++ ) f[k] = 0;
			puts("weights_");
			for( int k = 0 ; k < K ; k++ ) {
				if( expf( weights_[k] ) > 1e-10 ){
					f[k] = 1;
					cnt++;
					printf("%.5f\t",expf(weights_[k]));
				}
			}
			puts("\n");
			puts("means_");
			for( int k = 0 ; k < K ; k++ ){
				if( f[k] ){
					for( int d = 0; d < D ; d++ ){
						printf("%.3f\t", means_[d*K+k]);
					}
					puts("");
				}
			}
			puts("");
			puts("covariances_");
			for( int k = 0 ; k < K ; k++ ){
				if( f[k] ){
					for( int d = 0; d < D ; d++ ){
						printf("%.3f\t", 1 / powf( precision_cholesky_[d*K+k] , 2  ) );
					}
					puts("");
				}
			}

			puts("");

			printf("iteration:%d converged:%d log_likelihood:%.5f lower_bound:%.5f ",fin_iter,converged,log_likelihood,lower_bound);
			printf("init_class:%d finally_class:%d ",K,cnt);
		}

		void copy_param(){
			hipMemcpyAsync( means_ , dev_means_ , size_means_ , hipMemcpyDeviceToHost , st_means_ );
			hipMemcpyAsync( precision_cholesky_ , dev_precisions_ , size_precisions_ , hipMemcpyDeviceToHost , st_precisions_ );
			hipStreamSynchronize( st_means_ );
			hipStreamSynchronize( st_precisions_ );
		}

		//fit parameter to X


		bool fit( type *X , type tol , type *init_mean ){
			//std::cout << __FUNCTION__ << std::endl;
			double old_lower_bound = 0;

			START_TIME(FIT);

			START_TIME(INIT);
			_initialize(X , init_mean );
			END_TIME(INIT);

			for( int iter = 0 ; iter < n_iter ; iter++ ){
#ifdef DEBUG
				printf("iter:%d start\n", iter + 1);
#endif

				START_TIME(E_STEP);
				_e_step();

				END_TIME(E_STEP);

				//PRINT_CUDA_MAT_SCOPE( "e_step ato log resp" , dev_r_ , N , K , 3 );

				START_TIME(SR);
				double sum_resp;
				_compute_sum_resp(&sum_resp);
				END_TIME(SR);

				START_TIME(M_STEP);
				_m_step();
				END_TIME(M_STEP);

				START_TIME(LB);
				lower_bound = _compute_lower_bound( &sum_resp );
				END_TIME(LB);

#ifdef DEBUG
				printf("iter:%d log_likelihood:%.5f lower_bound:%.5f\n", iter + 1  , log_likelihood , lower_bound );
#endif

#ifndef TAKE_LL_EVERY_ITER
				//printf("iter:%d lower_bound:%.5f\n", iter + 1  , lower_bound );
#else
				printf("iter:%d log_likelihood:%.5f lower_bound:%.5f\n", iter + 1  , log_likelihood , lower_bound );
#endif
				if( abs( lower_bound - old_lower_bound ) < tol  || std::isnan(lower_bound) ){
					fin_iter = iter + 1;
					if(std::isnan(lower_bound)){
						converged = false;
						log_likelihood = std::numeric_limits<double>::quiet_NaN();
						printf("FAILED:lower_bound is nan\n");
					}
					else{
						START_TIME(LL);
						converged = true;
						get_last_log_likelihood( );
						printf("SUCCES:BGMM is converged\n");
						END_TIME(LL);
					}
					break;
				}
				old_lower_bound = lower_bound;
			}

			if( fin_iter == -1 ){
				converged = false;
				log_likelihood = std::numeric_limits<double>::quiet_NaN();
				fin_iter = n_iter;
				get_last_log_likelihood( );
				printf("FAILED:itertion reach n_iter");
			}

			START_TIME(COPY);
			copy_param();
			END_TIME(COPY);

			//std::cout << "end " << __FUNCTION__ << std::endl;
			END_TIME(FIT);
			return converged;
		}

		void output_clustering_result( char *res_file_name){

			_e_step();

			type *r;
			r = (type *)malloc(sizeof(type) * size_r_ );
			hipMemcpy( r , dev_r_ , size_r_ , hipMemcpyDeviceToHost );

			for( int n = 0 ; n < N ; n++ ){
				for( int k = 0 ; k < K ; k++ ){
					r[ k * N + n ] = expf( r[ k * N + n ] );
					//printf("%.1f " , r[ k * N + n ] );
					
				}
				//puts("");
			}

			FILE *res_file;
			if( ( res_file = fopen( res_file_name , "w" ) ) == NULL ){
				return;
			}

			int *res;
			res = (int *)malloc(sizeof(int) * N );

			for( ITR_SIZE n = 0 ; n < N ; n++ ){
				type M = 0;
				for( int k = 0 ; k < K ; k++ ){
					if( M < r[ k * N + n ] ){
						M = r[ k * N + n ];
						res[n] = k;
					}
				}
			}

			for( ITR_SIZE n = 0 ; n < N ; n++ ) fprintf( res_file , "%d," , res[n] );
			fprintf( res_file , "\n" );

			free(r);
			free(res);
		}

	private:

		void _compute_sum_resp( double *sum_resp ){
			substitute_zero <double> <<< 1 , 1 >>> (dev_1_);

			for( int k = 0 ; k < K ; k++ ){
				ITR_SIZE size = N;

				cpy_and_process_for_calc_sum_resp<type , threadNum_N > <<< grid_N , block_N >>> ( dev_r_ , dev_N_ , N , k);
				hipDeviceSynchronize();
				while( size > 1 ){
					dim3 s_block( threadNum_N , 1 , 1 );
					dim3 s_grid( size / s_block.x , 1 , 1 );
					if( size == threadNum_N )
						add_sum_resp< threadNum_N , n_type> <<< s_grid.x , s_block  >>> ( dev_N_, dev_1_, size );
					else
						get_sum_of_N< threadNum_N > <<< s_grid.x , s_block  >>> ( dev_N_ , dev_N_ , size );
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			hipMemcpyAsync( sum_resp , dev_1_ , sizeof(double) , hipMemcpyDeviceToHost  );
		}

		void _initialize( type *input_X , type *init_mean ){

#ifdef DEBUG
			printf("initalize start\n");
#endif


			set_value_matrix <type> <<< grid_N , block_N >>> ( dev_r_ , N , K , 1.0 / K );
			hipDeviceSynchronize();

			type xk[K*D];

#if USE_INIT_MEAN_FILE
			for( int k = 0 ; k < K ; k++ ) for( int d = 0 ; d < D ; d++ ) xk[d*K+k] = init_mean[d*K+k];
#else
			std::random_device rnd;
			std::mt19937 mt(rnd());
			for( int k = 0 ; k < K ; k++ ){
				//ITR_SIZE idx = N / K * k;
				ITR_SIZE idx = mt() % N;
				for( int d = 0 ; d < D ; d++ ){
					xk[d * K + k ] = input_X[ d * N + idx];
				}
			}
#endif


			hipMemcpyAsync( dev_means_ , &xk , size_means_ , hipMemcpyHostToDevice , st_means_ );

			//PRINT_CUDA_MAT("init mean" , dev_means_ , K , D );
			// update alpha , beta , nu
			type nk[K];
			for( int k = 0 ; k < K ; k++ ){
				nk[k] = N/K;
				//estimate weights
				weight_concentration_[k] = *weight_concentration_prior_ + nk[k];
				mean_precision_[k] = *mean_precision_prior_ + nk[k];
				degree_of_freedom_[k] = *degree_of_freedom_prior_ + nk[k];
			}
			hipMemcpyAsync( dev_K_ , &nk , sizeof(type) * K , hipMemcpyHostToDevice , st_KD_ );

			//debug
			//PRINT_HOST_VEC("nk" , nk , K ); 

			/*
				 PRINT_HOST_VEC("weight_concentration_" , weight_concentration_, K ); 
				 PRINT_HOST_VEC("mean_precision_" , mean_precision_ , K ); 
				 PRINT_HOST_VEC("degree_of_freedom_" , degree_of_freedom_ , K ); 
			 */

			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_ );


			//compute S
			hipStreamSynchronize(st_means_);
			pow_substitution_elements<type> <<< grid_K , block_K , 0 , st_means_ >>> ( dev_KD_2_ , dev_means_ , K , D  );



			hipblasHandle_t handle; //bottle neck
			hipblasCreate(&handle);

			START_TIME(F2);
			hipDeviceSynchronize();
			END_TIME(F2);

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_XD_ , N,
					&_b,
					dev_KD_ , K); //float

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_precisions_ , K); //float

			// estimate means
			for( int k = 0 ; k < K ; k++ ){
				for( int d = 0 ; d < D ; d++ ){
					means_[d*K+k]= ( ( *mean_precision_prior_ * mean_prior_[d]) + (nk[k] * xk[d*K+k] ) ) / mean_precision_[k];
				}
			}

			hipStreamSynchronize(st_means_);
			hipDeviceSynchronize();


			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_precisions_ , dev_means_ , K , D );

			//debug
			/*
				 PRINT_CUDA_MAT("avg_X2" , dev_KD_ , K , D );
				 PRINT_CUDA_MAT("avg_means_2" , dev_KD_2_ , K , D );
				 PRINT_CUDA_MAT("avg_X_means" , dev_precisions_ , K , D );
			 */

			//dev_KD <- S = covariance
			estimate_gaussian_covariance_diag <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_KD_2_ , dev_precisions_, dev_K_ , 1e-6, K , D );



			PRINT_CUDA_MAT("init nk", dev_K_ , 1 , K );
			PRINT_CUDA_MAT("init xk", dev_means_ , K , D );
			PRINT_CUDA_MAT("init sk", dev_KD_ , K , D );

			/*
				 PRINT_CUDA_MAT("degree_of_freedom_" , dev_degree_of_freedom_ , 1 , K );
				 PRINT_CUDA_MAT("mean_prior_" , dev_mean_prior_ , 1 , D );
				 PRINT_CUDA_MAT("mean_precision_" , dev_mean_precision_ , 1 , K );
				 PRINT_CUDA_MAT("covariance_prior_" , dev_covariance_prior, 1 , D );
			 */

			//estimate precisions
			//dev_precisions_ <- updated precisions_cholesky
			estimate_precisions <<< grid_K , block_K , 0 , st_KD_>>> ( dev_K_ , dev_means_ , dev_KD_ , dev_mean_prior_ , dev_precisions_ , dev_degree_of_freedom_ , dev_covariance_prior , dev_mean_precision_ ,  dev_mean_precision_prior , K , D ) ;

			//must execute after estimate precisions becase estimate pre need dev_means_ val = xk
			hipMemcpyAsync( dev_means_ , means_ , size_means_ , hipMemcpyHostToDevice , st_KD_ );

			/*
				 PRINT_CUDA_MAT("init means_" , dev_means_, K , D );
				 PRINT_CUDA_MAT("init precisions_cholesky" , dev_precisions_ , K , D );
			 */

			hipDeviceSynchronize();

			//for estimate_log_gaussian_prob
			compute_log_det_chol <<< grid_K , block_K , 0 , st_log_det_chol_ >>> ( dev_log_det_chol_ , dev_precisions_ , dev_degree_of_freedom_ ,  K , D );
#ifdef DEBUG
			printf("initalize end\n");
#endif

		}

		void get_last_log_likelihood( ){
			_estimate_weighted_log_prob_();
			hipDeviceSynchronize();
			get_log_likelihood();
		}

		void get_log_likelihood( ){
			get_sum_of_K <type, n_type > <<< grid_N , block_N >>> ( dev_r_ , dev_N_, N , K );
			hipDeviceSynchronize();

			//PRINT_CUDA_MAT_SCOPE("sum_of_K" , dev_N_ , N , 1 , 3 , 3  );

			ITR_SIZE size = N;
			dim3 block_S( threadNum_N , 1 , 1 );
			dim3 grid_S( size / block_S.x , 1 , 1 );
			while( size > 1 ){
				get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , N );
				hipDeviceSynchronize();
				size = size / threadNum_N  ;
			}
			n_type tmp;
			hipMemcpy( &tmp, dev_N_ , sizeof(n_type) , hipMemcpyDeviceToHost );
			hipDeviceSynchronize();
			//printf("sum resp:%lf\n" , tmp );
			log_likelihood = tmp / N ;
			return  ;
		}

		void _free_hst_memory(){
			hipFree(means_);
			hipFree(covariances_);
			hipFree(precision_cholesky_);
			hipFree(weights_);
			hipFree(mean_prior_);
			hipFree(degree_of_freedom_);
			hipFree(mean_precision_);
#ifdef INIT_MEAN_ON_HOST
			hipFree(covariance_prior_);
#endif
		}

		void _free_dev_memory(){
			hipFree(dev_X_);
			hipFree( dev_log_weights_ );
			hipFree( dev_r_ );
			hipFree( dev_precisions_ );
			hipFree( dev_K_ );
			hipFree( dev_XK_ );
			hipFree( dev_KD_  );
			hipFree( dev_KD_2_  );
			hipFree( dev_XD_ );
			hipFree( dev_log_det_chol_ );
			hipFree( dev_degree_of_freedom_ );
			hipFree( dev_mean_precision_ );
			hipFree( dev_N_ );
			hipFree( dev_mean_prior_ );  
			hipFree( dev_covariance_prior );
			hipFree( dev_mean_precision_prior );
			hipFree( dev_1_ );
		}

		void _destroy_stream(){
			hipStreamDestroy( st_X_ );
			hipStreamDestroy( st_X_ );
			hipStreamDestroy( st_lambda_);
			hipStreamDestroy( st_weights_);
			hipStreamDestroy( st_log_lambda_);
			hipStreamDestroy( st_log_weights_);
			hipStreamDestroy( st_r_ );
			hipStreamDestroy( st_precisions_);
			hipStreamDestroy( st_log_det_chol_);
			hipStreamDestroy( st_KD_ );
			hipStreamDestroy( st_means_ );
			hipStreamDestroy( st_degree_of_freedom_ );
			hipStreamDestroy( st_mean_precision_ );
			hipStreamDestroy( st_prior_ );
		}

		double _compute_lower_bound( double *sum_resp ){
			START_TIME(LOG_DET_CHOL);
			float _log_det_chol[K];
			compute_log_det_chol <<< grid_K , block_K , 0 , st_log_det_chol_ >>> ( dev_log_det_chol_ , dev_precisions_ , dev_degree_of_freedom_ ,  K , D );
			hipMemcpy( &_log_det_chol , dev_log_det_chol_ , size_log_det_chol , hipMemcpyDeviceToHost );
			hipDeviceSynchronize();
			END_TIME(LOG_DET_CHOL);

			//PRINT_HOST_VEC("log_det_chol" , _log_det_chol , K );
			START_TIME(LOW);

			for( int k = 0 ; k < K ; k++ ){
				_log_det_chol[k] = (double)_log_det_chol[k] - 0.5 * D * logf( degree_of_freedom_[k]);
			}
			type _sum_log_wishart = _compute_sum_log_wishart_norm( _log_det_chol );

			type _log_weight_norm = _log_dirichket_norm();

			type _sum_log_mean_precision = 0;

			for( int k = 0 ; k < K ; k++ ) _sum_log_mean_precision += logf(mean_precision_[k]);

			hipDeviceSynchronize();

			END_TIME(LOW);

#ifdef DEBUG
			printf("%.5f %.5f %.5f %.5f\n", *sum_resp , _sum_log_wishart , _log_weight_norm , 0.5 * D * _sum_log_mean_precision );
#endif

			return - *sum_resp - _sum_log_wishart - _log_weight_norm - 0.5 * D * _sum_log_mean_precision;
		}

		type _compute_sum_log_wishart_norm( type *log_det_precision_chol ){
			std::vector<double> log_wishart(K,0);
			for( unsigned int k = 0 ; k < K ; k++ ){
				double sum_gammaln_nu = 0.0;
				for( unsigned int d = 0 ; d < D ; d++ ) sum_gammaln_nu += lgamma( 0.5 * ( degree_of_freedom_[k] - d + 1 ) );
				log_wishart[k] =	- ( (double)degree_of_freedom_[k] * log_det_precision_chol[k] 
						+ (double)degree_of_freedom_[k] * D * 0.5 * logf(2.0)
						+ sum_gammaln_nu
						);
			}
			return (float)std::accumulate( log_wishart.begin() , log_wishart.end() , 0.0);
		}

		type _log_dirichket_norm(){
			double tmp = 0 , sum = 0;
			for( int k = 0; k < K ; k++ ){
				tmp += lgamma( weight_concentration_[k] );
				sum += weight_concentration_[k];
			}
			return (float) ( lgamma( sum ) - tmp );
		}

		/*** E step & its Kernels ***/
		void _e_step(){
			_estimate_weighted_log_prob_();


			START_TIME(LR);
			compute_log_resp <threadNum_N><<< grid_N , block_N >>> ( dev_K_ , dev_r_ , N , K ) ;
			hipDeviceSynchronize();
			END_TIME(LR);

			return;
		}

		void _estimate_weighted_log_prob_(){
			_estimate_log_gaussian_prob_();

			START_TIME(WEIGHT);
#ifndef DEBUG
			type tmp = boost::math::digamma( std::accumulate(  weight_concentration_.begin() , weight_concentration_.end() , 0.0 ) );
#else
			type tmp = std::accumulate(  weight_concentration_.begin() , weight_concentration_.end() , 0.0 ) ;
			if( tmp == 0 || tmp <= -1) tmp = 1;
			tmp = boost::math::digamma( tmp );
#endif
			for(unsigned int k = 0 ; k < K ; k++ ){
#ifdef DEBUG
				if( this->weight_concentration_[k] != 0 || this->weight_concentration_[k] <= -1 )
					this->weights_[k] = boost::math::digamma(this->weight_concentration_[k]) - tmp;
				else
					this->weights_[k] = boost::math::digamma( 1 ) - tmp;
#endif
				this->weights_[k] = boost::math::digamma(this->weight_concentration_[k]) - tmp;
			}

			hipMemcpyAsync( dev_log_weights_ , this->weights_ , size_weights_ , hipMemcpyHostToDevice  , st_log_weights_ );


			END_TIME(WEIGHT);

			START_TIME(LAMBDA);

			estimate_log_lambda_<type> <<< grid_K , block_K , sizeof(type) * K , st_log_lambda_ >>> ( dev_log_lambda_ , dev_degree_of_freedom_ ,  dev_mean_precision_, K , D );

#ifdef DEBUG
			PRINT_CUDA_MAT("log_weight" , dev_log_weights_ , 1 , K );
			PRINT_CUDA_MAT("log_lambda" , dev_log_lambda_ , 1 , K );
			PRINT_CUDA_MAT_SCOPE("log_gauss" , dev_r_ , N , K , 3 , 3 );
#endif


			hipDeviceSynchronize();
			END_TIME(LAMBDA);

			START_TIME(WLP);
			compute_weighted_log_prob  <<< grid_N , block_N , sizeof(double) * K >>> ( dev_r_ , dev_log_lambda_ , dev_log_weights_ , N , K );
			hipDeviceSynchronize();
			END_TIME(WLP);

#ifdef TAKE_LL_EVERY_ITER
			get_log_likelihood();
#endif

			/*
				 PRINT_CUDA_MAT("log_lambda" , dev_log_lambda_ , 1 , K );
				 PRINT_HOST_VEC ("weights_" , weights_, K );
				 PRINT_CUDA_MAT_SCOPE("weited_log_prob", dev_r_ , N , K , 5 , 5 );
			 */

			return;
		}

		void _estimate_log_gaussian_prob_( ){

			START_TIME(GAUSS);
			hipStreamSynchronize( st_precisions_ );
			hipStreamSynchronize( st_KD_) ;

			//get precisions
			pow_self_elements<type><<< grid_K , block_K , 0 , st_precisions_ >>> ( dev_precisions_ , K , D );

			/*
				 PRINT_CUDA_MAT("log_det_chol" , dev_log_det_chol_ , 1 , K );
				 PRINT_CUDA_MAT("means" , dev_means_ , K , D );
				 PRINT_CUDA_MAT("precisions" , dev_precisions_ , K , D );
			 */


			//get devK its kth componnent is sum k={0~K} means_(d,k) ** 2 * precisions_(d,k)
			pow_substitution_elements<type> <<< grid_K , block_K ,  0 , st_KD_ >>> ( dev_KD_ , dev_means_ , K , D );
			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_precisions_ , K , D );
			get_sum_of_2dim <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_K_ , dev_KD_ , K , D );
			hipDeviceSynchronize();

			//dev_r_ <- X ** 2  * precision^T

			//dev_XD_はコンストラクタ内で計算済み
			hipblasHandle_t handle;
			hipblasCreate(&handle);

			hipDeviceSynchronize();

			//m3
			hipblasSgemm(
					handle, HIPBLAS_OP_N , HIPBLAS_OP_T ,
					N , K , D ,
					&_a ,
					dev_XD_, N ,
					dev_precisions_ , K ,
					&_b,
					dev_r_ , N
					);
			hipDeviceSynchronize();

			/*
				 PRINT_CUDA_MAT_SCOPE("dev_XD" , dev_XD_ , N , D  , 3);
				 PRINT_CUDA_MAT("dev_precisions_" , dev_precisions_ , K , D );
			 */


			//dev_KD_ <- means◦ precisions
			hadamard_product_subtitution<type> <<< grid_K , block_K >>> ( dev_KD_ , dev_precisions_ , dev_means_ , K , D );

			//PRINT_CUDA_MAT("mean had pre " , dev_KD_ , K , D );


			//devXK <- devX * means◦ precisions
			hipDeviceSynchronize();

			//m2?
			hipblasSgemm(
					handle, HIPBLAS_OP_N , HIPBLAS_OP_T ,
					N , K , D,
					&_a ,
					dev_X_ , N,
					dev_KD_ , K,
					&_b,
					dev_XK_ , N
					);
			hipDeviceSynchronize();

			/*
				 PRINT_CUDA_MAT("log_det_chol" , dev_log_det_chol_ , 1 , K );
				 PRINT_CUDA_MAT("m1_2" , dev_K_ , 1 , K );
				 PRINT_CUDA_MAT_SCOPE("m2" , dev_XK_ , N , K , 3  );
				 PRINT_CUDA_MAT_SCOPE("m3" , dev_r_ , N , K , 3  );
			 */

			compute_log_gaussian_prob<type> <<< grid_N , block_N , type_width * (K) , st_degree_of_freedom_ >>> ( dev_r_ , dev_XK_ , dev_K_ ,   dev_log_det_chol_ , dev_degree_of_freedom_ , N , K , D);

			//PRINT_CUDA_MAT_SCOPE("e_step resp" , dev_r_ , N , K , 3 );


#if PROFILE
			hipDeviceSynchronize();
			END_TIME(GAUSS);
#endif
			return;
		}

		void _m_step(){
			//PRINT_CUDA_MAT_SCOPE("resp" , dev_r_ , N , K , 3 );

			START_TIME(NK);

			SUM(N,K,dev_r_,dev_K_,cpy_sum,fold_sum);

			//PRINT_CUDA_MAT("nk" , dev_K_ , 1 , K  );

			//compute_Nk
			for( ITR_SIZE k = 0 ; k < K ; k++ ){
				ITR_SIZE size = N;
				cpy_test <threadNum_N , type , n_type > <<< size / threadNum_N , threadNum_N >>> (  dev_r_ , dev_N_ ,N , k );
				size /= threadNum_N;
				hipDeviceSynchronize();

				// if N = Thread Num N
				if(size == 1 ){
					cpy <<<  1 , 1 >>> ( &dev_K_[k] , dev_N_ , 1 , 0  );
					continue;
				}

				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx< threadNum_N , n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_K_ , size , k);
					}
					else{
						substitute_sum_with_idx< threadNum_N , n_type , n_type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size, 0);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME(NK);
#endif


			//PRINT_CUDA_MAT("nk" , dev_K_ , 1 , K  );

			type nk[K];
			hipMemcpyAsync( &nk , dev_K_ , type_width * K , hipMemcpyDeviceToHost , st_X_  );

			START_TIME(MAKE_HANDLE);
			hipblasHandle_t handle;
			hipblasCreate(&handle);
			END_TIME(MAKE_HANDLE);

			START_TIME(XK);
			// compute Xbar
			// dev_means_ <- XBar = r^T * X
			hipDeviceSynchronize();
			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_means_ , K); //float

#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME(XK);
#endif

			START_TIME(PRI);
			hipStreamSynchronize(st_X_);
			// update alpha , beta , nu

			/*
				 float nk_sum = 0;
				 printf("nk\n");
				 for(int k = 0 ; k < K ; k++ ){
				 nk_sum += nk[k];
				 printf("%.5f " , nk[k]);
				 }
				 printf("nk_sum:%.5f\n",nk_sum);
			 */

			for( int k = 0 ; k < K ; k++ ){
				weight_concentration_[k] = *weight_concentration_prior_ + nk[k];
				mean_precision_[k] = *mean_precision_prior_ + nk[k];
				degree_of_freedom_[k] = *degree_of_freedom_prior_ + nk[k];
			}
			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_ );
			hipDeviceSynchronize();

			END_TIME(PRI);
			START_TIME2(XK);

			divide_and_check_XBar <<< grid_K , block_K , sizeof(float) * K , st_means_ >>> ( dev_means_ , dev_K_ , dev_mean_prior_ , K , D );
			type xk[D*K];
			hipMemcpyAsync( &xk , dev_means_ , size_means_ , hipMemcpyDeviceToHost , st_means_ );

#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME2(XK);
#endif

			START_TIME(SK);
			//compute S
			pow_substitution_elements<type> <<< grid_K , block_K , 0 , st_means_ >>> (dev_KD_2_ , dev_means_ , K , D  );

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_XD_ , N,
					&_b,
					dev_KD_ , K); //float

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_precisions_ , K); //float

#if PROFILE
			hipDeviceSynchronize();
			END_TIME(SK);
#endif

			START_TIME(MEAN);
			hipStreamSynchronize(st_means_);
			// compute means on hst
			for( int k = 0 ; k < K ; k++ ){
				if( nk[k] > 0 ){
					for( int d = 0 ; d < D ; d++ ){
						means_[d*K+k]= ( ( *mean_precision_prior_ * mean_prior_[d]) + (nk[k] * xk[d*K+k] ) ) / mean_precision_[k];
#ifdef DEBUG
						CHECK_ERROR("calc mean on hst" , means_[d*K+k]);
#endif
					}

				}else{
					for( int d = 0 ; d < D ; d++ )
						means_[d*K+k] = mean_prior_[k];
				}

			}

			END_TIME(MEAN);

			START_TIME2(SK);
			hipDeviceSynchronize();
			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_precisions_ , dev_means_ , K , D );

			//dev_KD <- S = covariance
			estimate_gaussian_covariance_diag <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_KD_2_ , dev_precisions_, dev_K_ , 1e-6, K , D );

#if PROFILE
			hipDeviceSynchronize();
			END_TIME2(SK);
#endif

			PRINT_CUDA_MAT("nk", dev_K_ , 1 , K );
			PRINT_CUDA_MAT("xk", dev_means_ , K , D );
			PRINT_CUDA_MAT("sk", dev_KD_ , K , D );

			/*
				 PRINT_HOST_VEC("weight_concentration_",weight_concentration_,K);
				 PRINT_HOST_VEC("mean_precision_",mean_precision_,K);
				 PRINT_CUDA_MAT("m_step mean_precision_" , dev_mean_precision_ , 1 , K );
				 PRINT_CUDA_MAT("m_step degree_of_freedom_" , dev_degree_of_freedom_ , 1 , K );
			 */


			START_TIME(PRECISION);
			//dev_precisions_ <- precision_cholesky
			estimate_precisions <<< grid_K , block_K , 0 , st_KD_>>> ( dev_K_ , dev_means_ , dev_KD_ , dev_mean_prior_ , dev_precisions_ , dev_degree_of_freedom_ , dev_covariance_prior , dev_mean_precision_ ,  dev_mean_precision_prior , K , D ) ;

			PRINT_CUDA_MAT("precision_cholesky" , dev_precisions_ , K , D );

#if PROFILE
			hipDeviceSynchronize();
			END_TIME(PRECISION);
#endif

			START_TIME(PTOE);

			//must do after beestimate_precisions
			hipMemcpyAsync( dev_means_ , means_ , size_means_ , hipMemcpyHostToDevice , st_KD_ );
			hipDeviceSynchronize();


			END_TIME(PTOE);
		}
};

int main(int argc , char *argv[]){
	ITR_SIZE N = 0, D = 0;
	TYPE *data , *tmp_init_mean;
	char* fileName;

#if USE_INIT_MEAN_FILE
	if( argc !=4  ) {
		printf("usage: %s <file> [cluster Num] [init mean file] \n", *argv);
		return 0;
	}
#else
	if( argc !=4    ){
		printf("usage: %s <file> [cluster Num] [clustering result filename] \n", *argv);
		return 0;
	}
#endif

	fileName = argv[1];
	ITR_SIZE K = atoi(argv[2]);

	char* clustering_result_file_name = argv[3];

	std::chrono::system_clock::time_point start_c, end_c;
	double time;

	data = loadMatrix<ITR_SIZE>( &D , &N , fileName );

	if( data == NULL ){
		printf("failed input data\n");
		return 1;
	}

	float *X;
	hipHostMalloc( &X , sizeof(float) * N * D );

	for( int d = 0 ; d < D ; d++ ){
		for( int n = 0 ; n < N ; n++ ){
			X[ d * N + n ] = data[ n * D + d];
		}
	}

#if USE_INIT_MEAN_FILE
	char* init_mean_fileName = argv[3];
	tmp_init_mean = loadMatrix<ITR_SIZE>( &D , &K , init_mean_fileName);
	float* init_mean = (float *)malloc(sizeof(float) * K * D);
	for( int d = 0 ; d < D ; d++ ){
		for( int k = 0 ; k < K ; k++ ){
			init_mean[d*K+k] = tmp_init_mean[k*D+d];
		}
	}
#else
	float *init_mean;
#endif

#ifdef PROF
	hipProfilerStart();
#endif

	//take time
	/*
		 hipEvent_t start, stop;
		 hipEventCreate(&start);
		 hipEventCreate(&stop);
		 hipEventRecord(start);
	 */

	start_c = std::chrono::system_clock::now();

	class vbgmm< float , double , THREAD_N > vb( X , N , K , D , 10000 );
	vb.fit(X , 1e-7 , init_mean );

	end_c = std::chrono::system_clock::now();
	time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end_c - start_c).count() / 1000.0);

	vb.output_clustering_result( clustering_result_file_name );

	start_c = std::chrono::system_clock::now();
	vb.free_memory_d();
	end_c = std::chrono::system_clock::now();

	time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end_c - start_c).count() / 1000.0);

	//hipEventRecord(stop);
	//hipEventSynchronize(stop);

	vb.print_param( );
	printf("time %lf[ms]\n", time);

	/*
		 float milliseconds = 0;
		 printf("cudaEvent time:%.5f\n", milliseconds );
		 hipEventElapsedTime(&milliseconds, start, stop);
	 */

	vb.free_memory_h();

#ifdef PROF
	hipProfilerStop();
#endif
	free(data);

	PRINT_TIMES();

	return 0;
}
