#include "hip/hip_runtime.h"

#include<chrono>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<float.h>
#include<time.h>
#include<iostream>
#include<vector>
#include<boost/math/special_functions/digamma.hpp>
#include<hip/hip_runtime.h>
#include<hipblas.h>

#ifdef PROF
#include<hip/hip_runtime_api.h>
#endif

#include"setting.hpp"
#include"cu_sum.hpp"
#include"profile.hpp"
#include"print.hpp"
#include"IO.hpp"
#include"xdisp.h"

#define B0  1                 /* 以下はBernoulli数 */
#define B1  (-1.0 / 2.0)
#define B2  ( 1.0 / 6.0)
#define B4  (-1.0 / 30.0)
#define B6  ( 1.0 / 42.0)
#define B8  (-1.0 / 30.0)
#define B10 ( 5.0 / 66.0)
#define B12 (-691.0 / 2730.0)
#define B14 ( 7.0 / 6.0)
#define B16 (-3617.0 / 510.0)

unsigned short ClusterNum = 2;

void GetImageFromStdin( TYPE *Data , const int D , bool TwiceScreenMode , unsigned int ColorAbstractionLevel ){
#if DEBUG
	printf("[start] %s\n" , __func__);
#endif

	unsigned int abs_level = pow( 2 , ColorAbstractionLevel - 1);

	for (int i=0; i<BITMAP; i++){
		for( int j = 0 ; j < 3 ; j++ ){
			Data[i * D + j ] = getchar() / abs_level;
		}
	}

	// 片目分の入力を破棄
	if( TwiceScreenMode )
		for (int i=0; i<BITMAP; i++)
			for( int j = 0 ; j < 3 ; j++ )
				getchar();

#if DEBUG
	printf("[end] %s\n" , __func__);
#endif
}

template<typename type>
__global__ void divide_arr( type *arr , unsigned int size , ITR_SIZE div_val  ){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	arr[idx] = arr[idx] / div_val;
}

template<typename type >
__global__
void set_value_matrix( type *mat , ITR_SIZE rows  , ITR_SIZE cols , const type val ){
	const ITR_SIZE row = blockIdx.x * blockDim.x + threadIdx.x;
	for( ITR_SIZE col = 0 ; col < cols ; col++ ){
		mat[ col*rows + row] = val;
	}
}

template <typename type, typename n_type ,  unsigned int threadNum >
__global__ void get_result( type *weigted_prob ,  unsigned int *DisplayData , ITR_SIZE DataSize , ITR_SIZE ClusterNum ){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int r,g,b;
	unsigned int result;

	if( idx < DataSize ){

		type res_val = -1;
		result = 0;
		for( int k = 0 ; k < ClusterNum ; k++ ){
			if(res_val < weigted_prob[k * DataSize + idx]){
				res_val = weigted_prob[ k * DataSize + idx ];
				result = k;
			}
		}

		r = (result % 3) * (255 / 3);
		g = (result % 4) * (255 / 4);
		b = (result % 5) * (255 / 5);
		DisplayData[idx] = ( r << 24 ) | ( g << 16 ) | ( b << 8 );
	}

}

__host__ __device__ float digamma(float xx ){
	float v, w;
	v = 0;
	while(xx < 100){
		v += 1/xx;
		xx++;
	}
	w = 1/(xx * xx);
	v += ((((((((B16 / 16) * w + (B14 /14)) * w + (B12 / 12)) * w + (B10 / 10)) * w + (B8 / 8)) * w + (B6 / 6)) * w + (B4 / 4))* w + (B2 / 2)) * w + 0.5 / xx;
	return log(xx) - v;
}

template<typename type>
__global__ void pow_self_elements( type *mat , const int rows , const int cols ){
	const int row = blockIdx.x * blockDim.x + threadIdx.x;

	if( row < rows )
		for( int col = 0 ; col < cols ; col++ ) mat[ row * cols + col ] = powf(mat[row * cols + col ] , 2);
}

template<typename type >
__global__ void compute_log_gaussian_prob( type *dev_r_ , type *dev_XK_, type *dev_K_ , type *dev_log_det_chol_ , type *dev_degree_of_freedom_ , const ITR_SIZE N , const int K , const int D ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ type share[];

	if( n < N ){

		for( int k = 0 ; k < K ; k++ )
			share[k] = dev_log_det_chol_[k] -  0.5 * D * logf(dev_degree_of_freedom_[k]);

		for( int k = 0 ; k < K ; k++ ){
			dev_r_[k*N+n] = -0.5 * ( D * logf( 2 * M_PI ) + ( dev_r_[k*N+n] - ( 2 * dev_XK_[k*N+n]) + dev_K_[k] ) ) + share[k];
		}
	}
}

__global__ void get_sum_of_2dim( float *devK , float *dev_KD_ , int K , int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		devK[k] = 0;
		for( int d = 0 ; d < D ; d++ ){
			devK[k] += dev_KD_[ d * K + k ];
		}
	}
}

template<typename type>
__global__ void compute_weighted_log_prob( type *dev_log_gauss_, type *dev_log_lambda_ , type *dev_log_weights_ , const ITR_SIZE N , const int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ double sum_lw[];

	if( n < N ){

		for( int k = 0 ; k < K ; k++ )
			sum_lw[k] = (double)dev_log_lambda_[k] + dev_log_weights_[k];

		for( int k = 0 ; k < K ; k++ )
			//dev_log_gauss_[ k * N + n ] = dev_log_gauss_[ k * N + n ] + dev_log_lambda_[k] + dev_log_weights_[k];
			dev_log_gauss_[ k * N + n ] = dev_log_gauss_[ k * N + n ] + sum_lw[k];
	}

}

template<unsigned const int threadNum>
__global__ void compute_log_resp( double *norm , float *resp , ITR_SIZE N , int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	if( n < N ){

		__shared__ float _max_of_kth[threadNum];


		_max_of_kth[threadIdx.x] = resp[n];
		for( int k = 1 ; k < K ; k++ ) _max_of_kth[threadIdx.x] = max( _max_of_kth[threadIdx.x] , resp[ k * N + n ] );
		float _sum = 0.0;
#ifdef FMATHA
		for( int k = 0 ; k < K ; k++ ) _sum += expf( __fsub_rd ( resp[k*N+n] , _max_of_kth[threadIdx.x] ));
		const float _logsumexp = logf( _sum ) + _max_of_kth[threadIdx.x];
		for( int k = 0 ; k < K ; k++ ) resp[k*N+n] = __fsub_rd ( resp[k*N+n] , _logsumexp );
#else
		for( int k = 0 ; k < K ; k++ ) _sum += expf( resp[k*N+n] - _max_of_kth[threadIdx.x] );
		const float _logsumexp = logf( _sum ) + _max_of_kth[threadIdx.x];
		norm[n] = _logsumexp;
		for( int k = 0 ; k < K ; k++ ) resp[k*N+n] =  resp[k*N+n] - _logsumexp ;
#endif
	}
}

template<typename type>
__global__ void pow_substitution_elements( type *to_mat , type *from_mat , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			to_mat[ d * K + k ] = powf( from_mat[d*K+k] , 2 );
		}
	}
}

template<typename type>
__global__ void hadamard_product_self( type *mat_a , type *mat_b , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			mat_a[d*K+k] = mat_a[d*K+k] * mat_b[d*K+k];
		}
	}
}

template<typename type>
__global__ void hadamard_product_subtitution( type *to_mat , type *from_a , type *from_b, const int  K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){
			to_mat[d * K + k] = from_a[d*K+k] * from_b[d*K+k];
		}
	}
}

//calc log Lambda for calc log resp
template<typename type>
__global__ void estimate_log_lambda_( float *log_lambda , float *degree_of_freedom_,  float *mean_precision_, const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ type share[];
	if ( k < K  ){
		double sum_digamma_nu_ = 0.0;

		if(threadIdx.x < K )
			share[k] = degree_of_freedom_[threadIdx.x] + 1;

		for( unsigned int d = 0 ; d < D ; d++ ){
			//digamma fuction
			sum_digamma_nu_ += logf(0.5*(share[k]-d) + 0.4849142) -  1 / ( 0.5*(share[k]-d) * 1.0271785 ) ;
		}
		log_lambda[k] = 0.5 * ( D * logf(2.0) + sum_digamma_nu_ - D / mean_precision_[k] ) ;
	}
}

template<const unsigned int threadNum , typename in_type , typename out_type>
__global__ void substitute_sum_with_idx( in_type *g_idata , out_type *g_odata , const ITR_SIZE N , const ITR_SIZE k){
	__shared__ out_type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	out_type mysum = (i < N) ? g_idata[i] : 0;
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x];
	sdata[tid] = mysum;
	__syncthreads();

	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x + k] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}

template<typename in_type , typename out_type >
__global__ void cpy( out_type *to , in_type *from, const ITR_SIZE N , const ITR_SIZE k){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	to[idx] = from[k * N + idx];
}

//double ver.
template<const unsigned int threadNum , typename in_type , typename out_type >
__global__ void cpy_test( in_type *g_idata, out_type *g_odata, const ITR_SIZE N , const unsigned int k ){
	__shared__ out_type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	out_type mysum = (i < N) ? g_idata[ k * N + i ] : 0;
	if ( i +  blockDim.x < N) mysum += g_idata[ ( k * N + i ) +  blockDim.x  ]  ;
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}



template<typename in_type , typename out_type>
__global__ void calc_for_get_covariance_prior( out_type *dev_N_ , in_type *dev_X_, in_type *dev_mean_prior_ , const ITR_SIZE N , const ITR_SIZE d){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;
	if( idx < N )
		dev_N_[idx] = pow( dev_X_[d * N + idx] - dev_mean_prior_[d]  , 2 );
}

template<typename out_type>
__global__ void calc_for_get_covariance_prior( out_type *dev_N_ , float *dev_X_, float *dev_mean_prior_ , const ITR_SIZE N , const ITR_SIZE d ){
	const ITR_SIZE idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < N )
		dev_N_[idx] = powf( dev_X_[d * N + idx] - dev_mean_prior_[d]  , 2 );

}

//2 type of cpy_and_process_for_calc_sum_resp
template<typename type , const unsigned int threadNum>
__global__ void cpy_and_process_for_calc_sum_resp( type *from , float *to , const ITR_SIZE N  , const int k ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;


	if( n < N ){

#ifdef FMATHB
		to[n] = (float) (__expf( from[k * N + n ] ) * (double)from[ k * N + n ]);
#else 
		to[n] = (float) (expf( from[k * N + n ] ) * (double)from[ k * N + n ]);
#endif

#ifdef FMATHG //非推奨
		from[k*N+n] = __expf(from[k*N+n]);
#else
		from[k*N+n] = expf(from[k*N+n]);
#endif
#ifdef DEBUG
		CHECK_ERROR("cpy_and_process_for_calc_sum_resp" , from[k*N+n]);
#endif
	}

}
template<typename type , const unsigned int threadNum>
__global__ void cpy_and_process_for_calc_sum_resp( type *from , double *to , const ITR_SIZE N  , const int k ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;


	if( n < N ){
		__shared__ double sdata[threadNum];
		sdata[threadIdx.x] = exp( (double)from[k * N + n ] );
		to[n] = sdata[threadIdx.x] * from[ k * N + n ];
		from[k*N+n] = sdata[threadIdx.x];
#ifdef DEBUG
		CHECK_ERROR("cpy_and_process_for_calc_sum_resp" , from[k*N+n]);
#endif
	}
}

__global__ void divide_and_check_XBar( float *xk , float *nk,  float *mean_prior_ , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ float share[];

	if(threadIdx.x < K ) share[threadIdx.x] = nk[threadIdx.x];
	__syncthreads();


	if( k < K ){
		for( int d = 0 ; d < D ; d++ ){

			if( share[k] > 1.0 )
				xk[d*K+k] = (double)xk[d*K+k] / share[k];
			else
				xk[d*K+k] = mean_prior_[k];

#ifdef DEBUG
			CHECK_ERROR("divide_and_check_XBar" , xk[d*K+k] );
#endif

		}
	}
}

__global__ void estimate_gaussian_covariance_diag( float *avg_X2 , float *avg_means_2 , float *avg_X_means , float *nk , float reg_covar , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	for( int d = 0 ; d < D ; d++ ){

		if( nk[k] > 0 )
			avg_X2[d*K+k] = ((double)avg_X2[d*K+k]/nk[k]) - 2 * ( (double)avg_X_means[d*K+k]/nk[k]) + avg_means_2[d*K+k];
		else
			avg_X2[d*K+k] = 1e-6;
		// prevent that S takeing negative value
		if( avg_X2[d*K+k] <= 0 ) {

#ifdef DEBUG
			printf("COLLECTION:estimate_gaussian_covariance_diag\n");
			CHECK_ERROR("estimate_gaussian_covariance_diag" , avg_X2[d*K+k]);
#endif
			avg_X2[d*K+k] = 1e-6;
		}
	}
}

__global__ void estimate_precisions( float *nk , float *xk, float *sk, float *mean_prior , float *precisions_cholesky , float *degree_of_freedom , float *covariance_prior , float *mean_precision ,  float *mean_precision_prior , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){


		if( nk[k] > 1.0 ){

			double diff[DIM] = {};
			for( int d = 0 ; d < D ; d++ ) diff[d] =  pow( xk[d*K+k] - mean_prior[d] , 2) ;

			for( int d = 0 ; d < D ; d++ ){
				diff[d] *= *mean_precision_prior / mean_precision[k];
				sk[d*K+k] += diff[d];
			}

			for( int d = 0 ; d < D ; d++ ){
				diff[d] = covariance_prior[d] + (double)nk[k] * sk[d*K+k];
			}

			//diff <- covariances
			for( int d = 0 ; d < D ; d++ ){
				diff[d] /= degree_of_freedom[k];
				precisions_cholesky[d*K+k] = 1 / sqrt(diff[d]);

#ifdef DEBUG
				CHECK_ERROR( "estimate_precisions" ,  precisions_cholesky[d*K+k]);
#endif
			}
		}
		else{
			for( int d = 0 ; d < D ; d++ )
				precisions_cholesky[d*K+k] = 1e-6;
		}
	}
}

template<const unsigned int threadNum , typename type>
__global__ void add_sum_resp( type *g_idata , double *sum_resp , const ITR_SIZE N){
	__shared__ type sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	type mysum = 0;
	if( i < N ){
		mysum = g_idata[i];
#ifdef DEBUG
		CHECK_ERROR("add_sum_resp" , mysum );
#endif
	}
	if( i + blockDim.x < N ){
#ifdef FMATHC
		mysum = __fadd_rd ( mysum , g_idata[i + blockDim.x ] );
#else
		mysum = mysum , g_idata[i + blockDim.x ];
#endif
#ifdef DEBUG
		CHECK_ERROR("add_sum_resp" , mysum );
#endif
	}
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
#ifdef FMATHC
			sdata[tid] = mysum = __fadd_rd( mysum , sdata[tid + s] );
#else
			sdata[tid] = mysum = mysum + sdata[tid + s];
#endif
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
#ifdef FMATHD
			mysum = __fadd_rd( mysum , __shfl_down_sync(FULL_MASK, mysum , offset) );
#else
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
#endif
		}
	}
	//sum_respが大きくなっている場合が多いので倍精度で加算
	if (tid == 0) *sum_resp = *sum_resp + (double)mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}


/*** 2type of get_sum_of_N ***/
//float ver.
template<const unsigned int threadNum>
__global__ void get_sum_of_N( float *g_idata , float *g_odata , const ITR_SIZE N ){
	__shared__ float sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	float mysum = (i < N) ? g_idata[i] : 0;
#ifdef FMATHE
	if (i + blockDim.x < N) mysum = __fadd_rd ( mysum , g_idata[i + blockDim.x] );
#else
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x]  ;
#endif
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
#ifdef FMATHE
			sdata[tid] = mysum = __fadd_rd ( mysum , sdata[tid + s] );
#else
			sdata[tid] = mysum = mysum + sdata[tid + s];
#endif
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
#ifdef FMATHF
			mysum = __fadd_rd ( mysum , __shfl_down_sync(FULL_MASK, mysum , offset) );
#else
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
#endif
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}
//double ver.
template<const unsigned int threadNum>
__global__ void get_sum_of_N( double *g_idata , double *g_odata , const ITR_SIZE N ){
	__shared__ double sdata[threadNum];
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 300)
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	double mysum = (i < N) ? g_idata[i] : 0;
	if (i + blockDim.x < N) mysum += g_idata[i + blockDim.x]  ;
	sdata[tid] = mysum;
	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
		if (tid < s) {
			sdata[tid] = mysum = mysum + sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid < 32) {
		if(blockDim.x >= 64) mysum += sdata[tid + 32];
		for (int offset = 32/2; offset>0; offset>>=1) {
			mysum += __shfl_down_sync(FULL_MASK, mysum , offset);
		}
	}
	if (tid == 0) g_odata[blockIdx.x] = mysum;
#else
#error "__shfl_down requires CUDA arch >= 300."
#endif
}

/*** end 2type of get_sum_of_N ***/

template<typename type>
__global__ void compute_log_det_chol( type *_log_det_chol , type *precision_chol, type *degree_of_freedom_ , const int K , const int D ){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if( k < K ){
		double sum = 0.0;
		for( int d = 0 ; d < D ; d++ ) sum += log( double( precision_chol[d*K+k])  );
		// for( int d = 0 ; d < D ; d++ ) sum += logf( precision_chol[k] ); なぜかバグらない
		_log_det_chol[k] = sum;
	}

}

template<typename type>
__global__ void substitute_zero( type *a ){
	*a = 0.0;
}


template<typename in_type , typename out_type >
__global__ void get_sum_of_K( in_type *resp  , out_type *dev_N_ , ITR_SIZE N , int K ){
	const ITR_SIZE n = blockIdx.x * blockDim.x + threadIdx.x;
	if( n < N ){
		float _max_of_kth = resp[n];
		for( int k = 1 ; k < K ; k++ ) _max_of_kth = max( _max_of_kth , resp[ k * N + n ] );
		double _sum = 0.0;
		for( int k = 0 ; k < K ; k++ ) _sum += exp( double(resp[k*N+n]) - _max_of_kth );
		dev_N_[n] = log(_sum) + _max_of_kth;
	}
}


template<typename type , typename n_type , unsigned int threadNum_N >
class vbgmm{

	public:
		size_t type_width;
		size_t size_means_ , size_covariances_ , size_precisions_ , size_precision_cholesky_ , size_weights_ , size_weight_concentration_ , size_mean_prior_ , size_degree_of_freedom_ , size_covariance_prior_ , size_mean_precision_ , size_gauss_ , size_X_ , size_lambda_ , size_r_ , size_log_det_chol ;

		//declare hst arrays
		type *means_ , *covariances_ , *precision_cholesky_ , *weights_ , *mean_prior_ , *degree_of_freedom_ ,  *mean_precision_ , *weight_concentration_prior_ , *mean_precision_prior_ , *degree_of_freedom_prior_  ;
		std::vector<type> weight_concentration_;
#if INIT_MEAN_ON_HOST
		type *covariance_prior_;
#endif

		//declare dev arrays
		type	*dev_means_ , *dev_log_resp_ , *dev_X_ , *dev_log_weights_ , *dev_r_ , *dev_log_lambda_ , *dev_precisions_ , *dev_K_, *dev_XK_ , *dev_KD_ , *dev_KD_2_ ,  *dev_log_det_chol_  , *dev_degree_of_freedom_  , *dev_XD_ , *dev_mean_precision_  ,
					*dev_mean_prior_ ,  *dev_covariance_prior ,   *dev_mean_precision_prior;
		n_type *dev_N_;
		double *dev_1_;
		unsigned int *dev_DisplayData;

		hipStream_t st_X_ , st_lambda_ , st_weights_ , st_log_lambda_ , st_log_weights_ , st_r_  , st_precisions_ , st_log_det_chol_ , st_KD_ , st_means_ , st_degree_of_freedom_ , st_mean_precision_  , st_prior_;

		bool converged = false;
		int n_iter , K , D , fin_iter = -1;
		ITR_SIZE N;

		//CUDA's parameter
		dim3 block_N = dim3( threadNum_N , 1 , 1) , grid_N;
		unsigned int threadNum_K;
		dim3 block_K , grid_K;
		double lower_bound , log_likelihood;

		const type _a = 1.0 , _b = 0.0;

		vbgmm(  const ITR_SIZE init_N , const unsigned int init_K , const unsigned int init_D , const unsigned int max_iter ){

			START_TIME(CONST);

			//std::cout << __FUNCTION__ << std::endl;
			N = init_N;
			K = init_K;
			D = init_D;

			if( N % block_N.x == 0 )
				grid_N = dim3( N / block_N.x  , 1 , 1 );
			else
				grid_N = dim3( N / block_N.x + 1 , 1 , 1 );

			threadNum_K = min ( K , 1024 );
			block_K = dim3(threadNum_K , 1 , 1 );
			grid_K = dim3( K / block_K.x  , 1 , 1);

			//set size of sum arrays
			type_width = sizeof(type);
			size_means_ = type_width * K * D;
			size_covariances_ = type_width * K * D;
			size_precisions_ = type_width * K * D;
			size_precision_cholesky_ = type_width * K * D;
			size_weights_ = type_width * K;
			size_weight_concentration_ = type_width * K;
			size_mean_prior_ = type_width * D;
			size_degree_of_freedom_ = type_width * K;
			size_covariance_prior_ = type_width * D;
			size_mean_precision_ = type_width * K;
			size_X_ = type_width * N * D;
			size_lambda_ = type_width * K;
			size_weights_ = type_width * K;
			size_r_ = type_width * N * K;
			size_log_det_chol = type_width * K;

			//malloc hst vec
			END_TIME(CONST);
			START_TIME(HOST_MALLOC);

			hipHostMalloc( &means_ , size_means_ );
			hipHostMalloc( &covariances_ , size_covariances_ );
			hipHostMalloc( &precision_cholesky_ , size_precision_cholesky_ );
			hipHostMalloc( &weights_ , size_weights_ );
			hipHostMalloc( &mean_prior_ , size_mean_prior_ );
			hipHostMalloc( &degree_of_freedom_ , size_degree_of_freedom_);
			hipHostMalloc( &mean_precision_ , size_mean_precision_);
			hipHostMalloc( &weight_concentration_prior_ , type_width );
			hipHostMalloc( &mean_precision_prior_ , type_width );
			hipHostMalloc( &degree_of_freedom_prior_ , type_width);
#if INIT_MEAN_ON_HOST
			hipHostMalloc( &covariance_prior_ , size_covariance_prior_);
#endif

			END_TIME(HOST_MALLOC);
			START_TIME(DEVICE_MALLOC);

			//malloc & init weight_concentration_ as 1/K
			weight_concentration_.assign( K , (float)1/K );

			//malloc dev vec
			hipMalloc( &dev_X_ , size_X_ );
			hipMalloc( &dev_log_lambda_ , size_lambda_ );
			hipMalloc( &dev_means_ , size_means_ );
			hipMalloc( &dev_log_weights_ , size_weights_ );
			hipMalloc( &dev_r_ , size_r_ );
			hipMalloc( &dev_precisions_ , size_precisions_ );
			hipMalloc( &dev_K_ , type_width * K);
			hipMalloc( &dev_XK_ , type_width * N * K );
			hipMalloc( &dev_KD_ , type_width * K * D );
			hipMalloc( &dev_KD_2_ , type_width * K * D );
			hipMalloc( &dev_XD_ , type_width * N * D );
			hipMalloc( &dev_log_det_chol_ , size_log_det_chol );
			hipMalloc( &dev_degree_of_freedom_ , size_degree_of_freedom_ );
			hipMalloc( &dev_mean_precision_ , size_mean_precision_ );
			hipMalloc( &dev_mean_prior_ , size_mean_prior_ );  
			hipMalloc( &dev_covariance_prior , size_covariance_prior_ );
			hipMalloc( &dev_mean_precision_prior , type_width );
			hipMalloc( &dev_1_ , sizeof(double) );

			hipMalloc( &dev_N_ , sizeof(n_type)* N );
			hipMalloc( &dev_DisplayData , sizeof( unsigned int  ) * N );

			hipStreamCreate( &st_X_ );
			hipStreamCreate( &st_lambda_);
			hipStreamCreate( &st_weights_);
			hipStreamCreate( &st_log_lambda_);
			hipStreamCreate( &st_log_weights_);
			hipStreamCreate( &st_r_ );
			hipStreamCreate( &st_precisions_);
			hipStreamCreate( &st_log_det_chol_);
			hipStreamCreate( &st_KD_ );
			hipStreamCreate( &st_means_ );
			hipStreamCreate( &st_degree_of_freedom_ );
			hipStreamCreate( &st_mean_precision_ );
			hipStreamCreate( &st_prior_ );

			END_TIME(DEVICE_MALLOC);
			START_TIME2(CONST);
			END_TIME2(CONST);

			//set number of iteration
			n_iter = max_iter;
		}

		void tranport_X( const type *input_X  ){
			//tranport X
			hipMemcpyAsync( dev_X_ , input_X , size_X_ , hipMemcpyHostToDevice , st_X_ );

			//dev_XD_をX ** 2 で初期化
			pow_substitution_elements <<< grid_N , block_N , 0 , st_X_ >>> ( dev_XD_ , dev_X_ , N , D );

			//PRINT_CUDA_IDX( "input data on gpu" , dev_X_ , N * D );
		}

		void set_prior(const type *input_X){
			//init weight_concentration_ as 1/K
			*weight_concentration_prior_ = (type)1/K; //alpha_0

			//init mean_precision_ as 1.0 & tranfer
			for( int k = 0 ; k < K ; k++ ) mean_precision_[k] = 1.0;
			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );

			//init mean_precision_prior_ as 1.0
			*mean_precision_prior_ = 1.0; // beta_0
			hipMemcpyAsync( dev_mean_precision_prior , mean_precision_prior_ , type_width , hipMemcpyHostToDevice , st_prior_ );

			//init degree_of_freedom_prior_ as D
			*degree_of_freedom_prior_ = D; //Nu_0

#if INIT_MEAN_ON_HOST
			//init mean_prior_ as X's mean
			for( int d = 0 ; d < D ; d++ ){
				double sum = 0.0;
				for( ITR_SIZE n = 0 ; n < N ; n++ ) sum += input_X[ d*N + n];
				mean_prior_[d] = (float)(sum/N);
			}
			hipMemcpyAsync( dev_mean_prior_ , mean_prior_ , size_mean_prior_ , hipMemcpyHostToDevice , st_prior_ );

			//init covariance_prior_ as X's cov
			for( int d = 0 ; d < D ; d++ ){
				double sum = 0.0;
				for( ITR_SIZE n = 0 ; n < N ; n++ ) sum += powf( input_X[d*N+n] - mean_prior_[d] , 2 );
				covariance_prior_[d] = (float)(sum/N);
			}
			hipMemcpyAsync( dev_covariance_prior , covariance_prior_ , size_covariance_prior_ , hipMemcpyHostToDevice , st_prior_ );
#else
			hipStreamSynchronize(st_X_);

			for( int d = 0 ; d < D ; d++ ){
				ITR_SIZE size = N;
				//cpy <type , n_type> <<< N / threadNum_N , threadNum_N >>> ( dev_N_ , dev_X_ , N , d);
				cpy_test <threadNum_N , type , n_type > <<< size / threadNum_N , threadNum_N >>> (  dev_X_ , dev_N_ ,N , d );
				size /= threadNum_N;
				hipDeviceSynchronize();

				if(size == 1 ){
					cpy <<<  1 , 1 >>> ( &dev_mean_prior_[d] , dev_N_ , 1 , 0  );
					continue;
				}

				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx<threadNum_N , n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_mean_prior_ , size , d);
					}
					else{
						get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			divide_arr<type> <<< D , ( D / threadNum_N) + 1 , 0 , st_means_ >>> ( dev_mean_prior_ , D ,  N );
			hipMemcpyAsync( mean_prior_ , dev_mean_prior_ , size_mean_prior_ , hipMemcpyDeviceToHost , st_prior_ );

			for( int d = 0 ; d < D ; d++ ){
				ITR_SIZE size = N;
				calc_for_get_covariance_prior <type , n_type > <<< N / threadNum_N , threadNum_N , 0 , st_means_>>> ( dev_N_ , dev_X_ , dev_mean_prior_ , N , d);
				hipDeviceSynchronize();
				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx<threadNum_N, n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_covariance_prior , size , d);
					}
					else{
						get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			divide_arr<type> <<< D , ( D / threadNum_N) + 1 >>> ( dev_covariance_prior , D ,  N );
#endif
			//init degree_of_freedom_ as 1.0
			for( int k = 0 ; k < K ; k++ ) degree_of_freedom_[k] = 1.0; // Nu
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_);


		}

		double predict( TYPE *X , unsigned int *DisplayData ){

			_e_step( );

			n_type log_prob_norm[N];

			// compute log likelihood
			double lower_bound = 0;
			hipMemcpy( log_prob_norm , dev_N_  , sizeof(n_type) * N , hipMemcpyDeviceToHost );
			for( int n = 0 ; n < N ; n++ ){
				lower_bound += log_prob_norm[n];
			}
			lower_bound /= N;

			get_result<type , n_type , threadNum_N > <<< grid_N , block_N >>> ( dev_r_ , dev_DisplayData , N , K  );
			hipMemcpy( DisplayData , dev_DisplayData , sizeof(unsigned int ) * N , hipMemcpyDeviceToHost );

			FILE *res_file;
			if( ( res_file = fopen( "DisplayData", "w" ) ) == NULL ){
				return 0;
			}

			for( int h = 0 ; h < HT ; h++ ){
				for( int w = 0 ; w < WD ; w++ ) fprintf( res_file , "%d" , DisplayData[h*WD+w] );
				fprintf(res_file , "\n");
			}

			fclose(res_file);

		

			return lower_bound;


			/*
			_e_step();

			type *r;
			r = (type *)malloc(sizeof(type) * size_r_ );
			hipMemcpy( r , dev_r_ , size_r_ , hipMemcpyDeviceToHost );

			for( int n = 0 ; n < N ; n++ ){
				for( int k = 0 ; k < K ; k++ ){
					r[ k * N + n ] = expf( r[ k * N + n ] );
					//printf("%.1f " , r[ k * N + n ] );
				}
				//puts("");
			}

			FILE *res_file;
			if( ( res_file = fopen( "res" , "w" ) ) == NULL ){
				exit(1);
			}

			int *res;
			res = (int *)malloc(sizeof(int) * N );

			for( ITR_SIZE n = 0 ; n < N ; n++ ){
				type M = 0;
				for( int k = 0 ; k < K ; k++ ){
					if( M < r[ k * N + n ] ){
						M = r[ k * N + n ];
						res[n] = k;
					}
				}
			}

			for( int h = 0 ; h < HT ; h++ ){
				for( int w = 0 ; w < WD ; w++ ) fprintf( res_file , "%d" , res[h*WD+w] );
				fprintf(res_file , "\n");
			}

			unsigned int re,g,b;
			for(int idx = 0 ; idx < BITMAP ; idx++ ){
				re = (res[idx] % 3) * (255 / 3);
				g = (res[idx] % 4) * (255 / 4);
				b = (res[idx] % 5) * (255 / 5);
				DisplayData[idx] = ( re << 24 ) | ( g << 16 ) | ( b << 8 );
			}

			free(r);
			free(res);

			return 1.0;
			*/
		}



		//free arrays that were malloced
		void free_memory_d(){
			START_TIME(FREE);
			_free_dev_memory();
			_destroy_stream();
			END_TIME(FREE);
		}

		void free_memory_h(){
			_free_hst_memory();
		}

		void print_param(){
			int cnt = 0;
			bool f[K];
			for( int k = 0 ; k < K ; k++ ) f[k] = 0;
			puts("weights_");
			for( int k = 0 ; k < K ; k++ ) {
				if( expf( weights_[k] ) > 1e-10 ){
					f[k] = 1;
					cnt++;
					printf("%.5f\t",expf(weights_[k]));
				}
			}
			puts("\n");
			puts("means_");
			for( int k = 0 ; k < K ; k++ ){
				if( f[k] ){
					for( int d = 0; d < D ; d++ ){
						printf("%.3f\t", means_[d*K+k]);
					}
					puts("");
				}
			}
			puts("");
			puts("covariances_");
			for( int k = 0 ; k < K ; k++ ){
				if( f[k] ){
					for( int d = 0; d < D ; d++ ){
						printf("%.3f\t", 1 / powf( precision_cholesky_[d*K+k] , 2  ) );
					}
					puts("");
				}
			}

			puts("");

			printf("iteration:%d converged:%d log_likelihood:%.5f lower_bound:%.5f ",fin_iter,converged,log_likelihood,lower_bound);
			printf("init_class:%d finally_class:%d ",K,cnt);
		}

		void copy_param(){
			hipMemcpyAsync( means_ , dev_means_ , size_means_ , hipMemcpyDeviceToHost , st_means_ );
			hipMemcpyAsync( precision_cholesky_ , dev_precisions_ , size_precisions_ , hipMemcpyDeviceToHost , st_precisions_ );
			hipStreamSynchronize( st_means_ );
			hipStreamSynchronize( st_precisions_ );
		}

		//fit parameter to X


		bool fit( type *X , type tol ){
			//std::cout << __FUNCTION__ << std::endl;
			double old_lower_bound = 0;

			START_TIME(FIT);

			START_TIME(INIT);
			_initialize(X  );
			END_TIME(INIT);

			for( int iter = 0 ; iter < n_iter ; iter++ ){
				//printf("iter:%d start\n", iter + 1);

				START_TIME(E_STEP);
				_e_step();

				END_TIME(E_STEP);

				//PRINT_CUDA_MAT_SCOPE( "e_step ato log resp" , dev_r_ , N , K , 3 );

				START_TIME(SR);
				double sum_resp;
				_compute_sum_resp(&sum_resp);
				END_TIME(SR);

				START_TIME(M_STEP);
				_m_step();
				END_TIME(M_STEP);

				START_TIME(LB);
				lower_bound = _compute_lower_bound( &sum_resp );
				END_TIME(LB);

				//get precisions
				pow_self_elements<type><<< grid_K , block_K , 0 , st_precisions_ >>> ( dev_precisions_ , K , D );

#ifdef DEBUG
				printf("iter:%d log_likelihood:%.5f lower_bound:%.5f\n", iter + 1  , log_likelihood , lower_bound );
#endif

#ifndef TAKE_LL_EVERY_ITER
				//printf("iter:%d lower_bound:%.5f\n", iter + 1  , lower_bound );
#else
				printf("iter:%d log_likelihood:%.5f lower_bound:%.5f\n", iter + 1  , log_likelihood , lower_bound );
#endif
				if( abs( lower_bound - old_lower_bound ) < tol  || std::isnan(lower_bound) ){
					fin_iter = iter + 1;
					if(std::isnan(lower_bound)){
						converged = false;
						log_likelihood = std::numeric_limits<double>::quiet_NaN();
						printf("FAILED:lower_bound is nan\n");
					}
					else{
						START_TIME(LL);
						converged = true;
						get_last_log_likelihood( );
						printf("SUCCES:BGMM is converged\n");
						END_TIME(LL);
					}
					break;
				}
				old_lower_bound = lower_bound;
			}

			if( fin_iter == -1 ){
				converged = false;
				log_likelihood = std::numeric_limits<double>::quiet_NaN();
				fin_iter = n_iter;
				get_last_log_likelihood( );
				printf("FAILED:itertion reach n_iter");
			}

			START_TIME(COPY);
			copy_param();
			END_TIME(COPY);

			//std::cout << "end " << __FUNCTION__ << std::endl;
			END_TIME(FIT);
			return converged;
		}

		void output_clustering_result( char *res_file_name){

			_e_step();

			type *r;
			r = (type *)malloc(sizeof(type) * size_r_ );
			hipMemcpy( r , dev_r_ , size_r_ , hipMemcpyDeviceToHost );

			for( int n = 0 ; n < N ; n++ ){
				for( int k = 0 ; k < K ; k++ ){
					r[ k * N + n ] = expf( r[ k * N + n ] );
					//printf("%.1f " , r[ k * N + n ] );
				}
				//puts("");
			}

			FILE *res_file;
			if( ( res_file = fopen( res_file_name , "w" ) ) == NULL ){
				return;
			}

			int *res;
			res = (int *)malloc(sizeof(int) * N );

			for( ITR_SIZE n = 0 ; n < N ; n++ ){
				type M = 0;
				for( int k = 0 ; k < K ; k++ ){
					if( M < r[ k * N + n ] ){
						M = r[ k * N + n ];
						res[n] = k;
					}
				}
			}

			/*
				 for( ITR_SIZE n = 0 ; n < N ; n++ ) fprintf( res_file , "%d," , res[n] );
				 fprintf( res_file , "\n" );
			 */


			for( int h = 0 ; h < HT ; h++ ){
				for( int w = 0 ; w < WD ; w++ ) fprintf( res_file , "%d" , res[h*WD+w] );
				fprintf(res_file , "\n");
			}

			free(r);
			free(res);
		}

	private:

		void _compute_sum_resp( double *sum_resp ){
			//substitute_zero <double> <<< 1 , 1 >>> (dev_1_);

			//PRINT_CUDA_IDX( "before SUM_exp" , dev_r_ , N * K );
			SUM_exp(N,K,dev_r_,dev_K_,cpy_sum_exp,fold_sum);
			//PRINT_CUDA_IDX( "after SUM_exp" , dev_r_ , N * K );
			float tmp_k[K];

			hipMemcpy( tmp_k , dev_K_ , sizeof(type) * K , hipMemcpyDeviceToHost );
			for( int k = 0 ; k < K ; k++ ) *sum_resp +=tmp_k[k];

			/*

				 for( int k = 0 ; k < K ; k++ ){
				 ITR_SIZE size = N;

				 cpy_and_process_for_calc_sum_resp<type , threadNum_N > <<< grid_N , block_N >>> ( dev_r_ , dev_N_ , N , k);
				 hipDeviceSynchronize();
				 while( size > 1 ){
				 dim3 s_block( threadNum_N , 1 , 1 );
				 dim3 s_grid( size / s_block.x , 1 , 1 );
				 if( size == threadNum_N )
				 add_sum_resp< threadNum_N , n_type> <<< s_grid.x , s_block  >>> ( dev_N_, dev_1_, size );
				 else
				 get_sum_of_N< threadNum_N > <<< s_grid.x , s_block  >>> ( dev_N_ , dev_N_ , size );
				 hipDeviceSynchronize();
				 size = size / threadNum_N ;
				 }
				 }
			 */

			//hipMemcpyAsync( sum_resp , dev_1_ , sizeof(double) , hipMemcpyDeviceToHost  );
		}

		void _initialize( type *input_X ){

#ifdef DEBUG
			printf("initalize start\n");
#endif

			set_value_matrix <type> <<< grid_N , block_N >>> ( dev_r_ , N , K , 1.0 / K );
			hipDeviceSynchronize();

			type xk[K*D];

			std::random_device rnd;
			std::mt19937 mt(rnd());
			for( int k = 0 ; k < K ; k++ ){
				//ITR_SIZE idx = N / K * k;
				ITR_SIZE idx = mt() % N;
				for( int d = 0 ; d < D ; d++ ){
					xk[d * K + k ] = input_X[ d * N + idx];
				}
			}

			hipMemcpyAsync( dev_means_ , &xk , size_means_ , hipMemcpyHostToDevice , st_means_ );

			//PRINT_CUDA_MAT("init mean" , dev_means_ , K , D );
			// update alpha , beta , nu
			type nk[K];
			for( int k = 0 ; k < K ; k++ ){
				nk[k] = N/K;
				//estimate weights
				weight_concentration_[k] = *weight_concentration_prior_ + nk[k];
				mean_precision_[k] = *mean_precision_prior_ + nk[k];
				degree_of_freedom_[k] = *degree_of_freedom_prior_ + nk[k];
			}
			hipMemcpyAsync( dev_K_ , &nk , sizeof(type) * K , hipMemcpyHostToDevice , st_KD_ );

			//debug
			//PRINT_HOST_VEC("nk" , nk , K ); 

			/*
				 PRINT_HOST_VEC("weight_concentration_" , weight_concentration_, K ); 
				 PRINT_HOST_VEC("mean_precision_" , mean_precision_ , K ); 
				 PRINT_HOST_VEC("degree_of_freedom_" , degree_of_freedom_ , K ); 
			 */

			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_ );


			//compute S
			hipStreamSynchronize(st_means_);
			pow_substitution_elements<type> <<< grid_K , block_K , 0 , st_means_ >>> ( dev_KD_2_ , dev_means_ , K , D  );

			hipblasHandle_t handle; //bottle neck
			hipblasCreate(&handle);

			START_TIME(F2);
			hipDeviceSynchronize();
			END_TIME(F2);


			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_XD_ , N,
					&_b,
					dev_KD_ , K); //float

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_precisions_ , K); //float

			// estimate means
			for( int k = 0 ; k < K ; k++ ){
				for( int d = 0 ; d < D ; d++ ){
					means_[d*K+k]= ( ( *mean_precision_prior_ * mean_prior_[d]) + (nk[k] * xk[d*K+k] ) ) / mean_precision_[k];
				}
			}

			hipStreamSynchronize(st_means_);
			hipDeviceSynchronize();


			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_precisions_ , dev_means_ , K , D );

			//debug
			/*
				 PRINT_CUDA_MAT("avg_X2" , dev_KD_ , K , D );
				 PRINT_CUDA_MAT("avg_means_2" , dev_KD_2_ , K , D );
				 PRINT_CUDA_MAT("avg_X_means" , dev_precisions_ , K , D );
			 */

			//dev_KD <- S = covariance
			estimate_gaussian_covariance_diag <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_KD_2_ , dev_precisions_, dev_K_ , 1e-6, K , D );


			PRINT_CUDA_MAT("init nk", dev_K_ , 1 , K );
			PRINT_CUDA_MAT("init xk", dev_means_ , K , D );
			PRINT_CUDA_MAT("init sk", dev_KD_ , K , D );

			PRINT_CUDA_MAT("degree_of_freedom_" , dev_degree_of_freedom_ , 1 , K );
			PRINT_CUDA_MAT("mean_prior_" , dev_mean_prior_ , 1 , D );
			PRINT_CUDA_MAT("mean_precision_" , dev_mean_precision_ , 1 , K );
			PRINT_CUDA_MAT("covariance_prior_" , dev_covariance_prior, 1 , D );

			//estimate precisions
			//dev_precisions_ <- updated precisions_cholesky
			estimate_precisions <<< grid_K , block_K , 0 , st_KD_>>> ( dev_K_ , dev_means_ , dev_KD_ , dev_mean_prior_ , dev_precisions_ , dev_degree_of_freedom_ , dev_covariance_prior , dev_mean_precision_ ,  dev_mean_precision_prior , K , D ) ;
			pow_self_elements<type><<< grid_K , block_K , 0 , st_KD_ >>> ( dev_precisions_ , K , D );
			//must execute after estimate precisions becase estimate pre need dev_means_ val = xk
			hipMemcpyAsync( dev_means_ , means_ , size_means_ , hipMemcpyHostToDevice , st_KD_ );

			PRINT_CUDA_MAT("init means_" , dev_means_, K , D );
			PRINT_CUDA_MAT("init precisions_cholesky" , dev_precisions_ , K , D );

			hipDeviceSynchronize();

			//for estimate_log_gaussian_prob
			compute_log_det_chol <<< grid_K , block_K , 0 , st_log_det_chol_ >>> ( dev_log_det_chol_ , dev_precisions_ , dev_degree_of_freedom_ ,  K , D );
#ifdef DEBUG
			printf("initalize end\n");
#endif

		}

		void get_last_log_likelihood( ){
			_estimate_weighted_log_prob_();
			hipDeviceSynchronize();
			get_log_likelihood();
		}

		void get_log_likelihood( ){
			get_sum_of_K <type, n_type > <<< grid_N , block_N >>> ( dev_r_ , dev_N_, N , K );
			hipDeviceSynchronize();

			//PRINT_CUDA_MAT_SCOPE("sum_of_K" , dev_N_ , N , 1 , 3 , 3  );

			ITR_SIZE size = N;
			dim3 block_S( threadNum_N , 1 , 1 );
			dim3 grid_S( size / block_S.x , 1 , 1 );
			while( size > 1 ){
				get_sum_of_N<threadNum_N> <<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , N );
				hipDeviceSynchronize();
				size = size / threadNum_N  ;
			}
			n_type tmp;
			hipMemcpy( &tmp, dev_N_ , sizeof(n_type) , hipMemcpyDeviceToHost );
			hipDeviceSynchronize();
			//printf("sum resp:%lf\n" , tmp );
			log_likelihood = tmp / N ;
			return  ;
		}

		void _free_hst_memory(){
			hipFree(means_);
			hipFree(covariances_);
			hipFree(precision_cholesky_);
			hipFree(weights_);
			hipFree(mean_prior_);
			hipFree(degree_of_freedom_);
			hipFree(mean_precision_);
#if INIT_MEAN_ON_HOST
			hipFree(covariance_prior_);
#endif
		}

		void _free_dev_memory(){
			hipFree(dev_X_);
			hipFree( dev_log_weights_ );
			hipFree( dev_r_ );
			hipFree( dev_precisions_ );
			hipFree( dev_K_ );
			hipFree( dev_XK_ );
			hipFree( dev_KD_  );
			hipFree( dev_KD_2_  );
			hipFree( dev_XD_ );
			hipFree( dev_log_det_chol_ );
			hipFree( dev_degree_of_freedom_ );
			hipFree( dev_mean_precision_ );
			hipFree( dev_N_ );
			hipFree( dev_mean_prior_ );  
			hipFree( dev_covariance_prior );
			hipFree( dev_mean_precision_prior );
			hipFree( dev_1_ );
			hipFree( dev_DisplayData );
		}

		void _destroy_stream(){
			hipStreamDestroy( st_X_ );
			hipStreamDestroy( st_X_ );
			hipStreamDestroy( st_lambda_);
			hipStreamDestroy( st_weights_);
			hipStreamDestroy( st_log_lambda_);
			hipStreamDestroy( st_log_weights_);
			hipStreamDestroy( st_r_ );
			hipStreamDestroy( st_precisions_);
			hipStreamDestroy( st_log_det_chol_);
			hipStreamDestroy( st_KD_ );
			hipStreamDestroy( st_means_ );
			hipStreamDestroy( st_degree_of_freedom_ );
			hipStreamDestroy( st_mean_precision_ );
			hipStreamDestroy( st_prior_ );
		}

		double _compute_lower_bound( double *sum_resp ){
			START_TIME(LOG_DET_CHOL);
			float _log_det_chol[K];
			compute_log_det_chol <<< grid_K , block_K , 0 , st_log_det_chol_ >>> ( dev_log_det_chol_ , dev_precisions_ , dev_degree_of_freedom_ ,  K , D );
			hipMemcpy( &_log_det_chol , dev_log_det_chol_ , size_log_det_chol , hipMemcpyDeviceToHost );
			hipDeviceSynchronize();
			END_TIME(LOG_DET_CHOL);

			//PRINT_HOST_VEC("log_det_chol" , _log_det_chol , K );
			START_TIME(LOW);

			for( int k = 0 ; k < K ; k++ ){
				_log_det_chol[k] = (double)_log_det_chol[k] - 0.5 * D * logf( degree_of_freedom_[k]);
			}
			type _sum_log_wishart = _compute_sum_log_wishart_norm( _log_det_chol );

			type _log_weight_norm = _log_dirichket_norm();

			type _sum_log_mean_precision = 0;

			for( int k = 0 ; k < K ; k++ ) _sum_log_mean_precision += logf(mean_precision_[k]);

			hipDeviceSynchronize();

			END_TIME(LOW);

#ifdef DEBUG
			printf("%.5f %.5f %.5f %.5f\n", *sum_resp , _sum_log_wishart , _log_weight_norm , 0.5 * D * _sum_log_mean_precision );
#endif

			return - *sum_resp - _sum_log_wishart - _log_weight_norm - 0.5 * D * _sum_log_mean_precision;
		}

		type _compute_sum_log_wishart_norm( type *log_det_precision_chol ){
			std::vector<double> log_wishart(K,0);
			for( unsigned int k = 0 ; k < K ; k++ ){
				double sum_gammaln_nu = 0.0;
				for( unsigned int d = 0 ; d < D ; d++ ) sum_gammaln_nu += lgamma( 0.5 * ( degree_of_freedom_[k] - d + 1 ) );
				log_wishart[k] =	- ( (double)degree_of_freedom_[k] * log_det_precision_chol[k] 
						+ (double)degree_of_freedom_[k] * D * 0.5 * logf(2.0)
						+ sum_gammaln_nu
						);
			}
			return (float)std::accumulate( log_wishart.begin() , log_wishart.end() , 0.0);
		}

		type _log_dirichket_norm(){
			double tmp = 0 , sum = 0;
			for( int k = 0; k < K ; k++ ){
				tmp += lgamma( weight_concentration_[k] );
				sum += weight_concentration_[k];
			}
			return (float) ( lgamma( sum ) - tmp );
		}

		/*** E step & its Kernels ***/
		void _e_step(){

			_estimate_weighted_log_prob_();


			START_TIME(LR);

			// dev_r  <- log likelihood
			// dev_N_ <- log prob norm
			compute_log_resp <threadNum_N><<< grid_N , block_N >>> ( dev_N_ , dev_r_ , N , K );

			PRINT_CUDA_MAT_SCOPE("Log Resp" , dev_r_ , N , K , 3 );

			hipDeviceSynchronize();
			END_TIME(LR);

			return;
		}

		void _estimate_weighted_log_prob_(){
			_estimate_log_gaussian_prob_();

			START_TIME(WEIGHT);
#ifndef DEBUG
			type tmp = boost::math::digamma( std::accumulate(  weight_concentration_.begin() , weight_concentration_.end() , 0.0 ) );
#else
			type tmp = std::accumulate(  weight_concentration_.begin() , weight_concentration_.end() , 0.0 ) ;
			if( tmp == 0 || tmp <= -1) tmp = 1;
			tmp = boost::math::digamma( tmp );
#endif
			for(unsigned int k = 0 ; k < K ; k++ ){
#ifdef DEBUG
				if( this->weight_concentration_[k] != 0 || this->weight_concentration_[k] <= -1 )
					this->weights_[k] = boost::math::digamma(this->weight_concentration_[k]) - tmp;
				else
					this->weights_[k] = boost::math::digamma( 1 ) - tmp;
#endif
				this->weights_[k] = boost::math::digamma(this->weight_concentration_[k]) - tmp;
			}

			hipMemcpyAsync( dev_log_weights_ , this->weights_ , size_weights_ , hipMemcpyHostToDevice  , st_log_weights_ );


			END_TIME(WEIGHT);

			START_TIME(LAMBDA);

			estimate_log_lambda_<type> <<< grid_K , block_K , sizeof(type) * K , st_log_lambda_ >>> ( dev_log_lambda_ , dev_degree_of_freedom_ ,  dev_mean_precision_, K , D );


			hipDeviceSynchronize();
			END_TIME(LAMBDA);

			START_TIME(WLP);
			compute_weighted_log_prob  <<< grid_N , block_N , sizeof(double) * K >>> ( dev_r_ , dev_log_lambda_ , dev_log_weights_ , N , K );
			hipDeviceSynchronize();
			END_TIME(WLP);

#ifdef TAKE_LL_EVERY_ITER
			get_log_likelihood();
#endif

			/*
				 PRINT_CUDA_MAT("log_lambda" , dev_log_lambda_ , 1 , K );
				 PRINT_HOST_VEC ("weights_" , weights_, K );
				 PRINT_CUDA_MAT_SCOPE("weited_log_prob", dev_r_ , N , K , 5 , 5 );
			 */

			return;
		}

		void _estimate_log_gaussian_prob_( ){

			START_TIME(GAUSS);
			hipStreamSynchronize( st_precisions_ );
			hipStreamSynchronize( st_KD_) ;


			PRINT_CUDA_MAT("log_det_chol" , dev_log_det_chol_ , 1 , K );
			PRINT_CUDA_MAT("means" , dev_means_ , K , D );
			PRINT_CUDA_MAT("precisions" , dev_precisions_ , K , D );

			//get devK its kth componnent is sum k={0~K} means_(d,k) ** 2 * precisions_(d,k)
			pow_substitution_elements<type> <<< grid_K , block_K ,  0 , st_KD_ >>> ( dev_KD_ , dev_means_ , K , D );
			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_precisions_ , K , D );
			get_sum_of_2dim <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_K_ , dev_KD_ , K , D );
			hipDeviceSynchronize();

			//dev_r_ <- X ** 2  * precision^T

			//dev_XD_はコンストラクタ内で計算済み
			hipblasHandle_t handle;
			hipblasCreate(&handle);

			hipDeviceSynchronize();

			//m3
			hipblasSgemm(
					handle, HIPBLAS_OP_N , HIPBLAS_OP_T ,
					N , K , D ,
					&_a ,
					dev_XD_, N ,
					dev_precisions_ , K ,
					&_b,
					dev_r_ , N
					);
			hipDeviceSynchronize();

			PRINT_CUDA_MAT_SCOPE("dev_XD" , dev_XD_ , N , D  , 3);
			PRINT_CUDA_MAT("dev_precisions_" , dev_precisions_ , K , D );
			PRINT_CUDA_MAT_SCOPE("dev_r_" , dev_r_ , N , K , 3 );

			//dev_KD_ <- means◦ precisions
			hadamard_product_subtitution<type> <<< grid_K , block_K >>> ( dev_KD_ , dev_precisions_ , dev_means_ , K , D );

			//PRINT_CUDA_MAT("mean had pre " , dev_KD_ , K , D );

			//devXK <- devX * means◦ precisions
			hipDeviceSynchronize();

			//m2?
			hipblasSgemm(
					handle, HIPBLAS_OP_N , HIPBLAS_OP_T ,
					N , K , D,
					&_a ,
					dev_X_ , N,
					dev_KD_ , K,
					&_b,
					dev_XK_ , N
					);
			hipDeviceSynchronize();


			PRINT_CUDA_MAT("log_det_chol" , dev_log_det_chol_ , 1 , K );
			PRINT_CUDA_MAT("m1_2" , dev_K_ , 1 , K );
			PRINT_CUDA_MAT_SCOPE("m2" , dev_XK_ , N , K , 3  );
			PRINT_CUDA_MAT_SCOPE("m3" , dev_r_ , N , K , 3  );

			compute_log_gaussian_prob<type> <<< grid_N , block_N , type_width * (K) , st_degree_of_freedom_ >>> ( dev_r_ , dev_XK_ , dev_K_ ,   dev_log_det_chol_ , dev_degree_of_freedom_ , N , K , D);

			//PRINT_CUDA_MAT_SCOPE("e_step resp" , dev_r_ , N , K , 3 );


#if PROFILE
			hipDeviceSynchronize();
			END_TIME(GAUSS);
#endif
			return;
		}

		void _m_step(){
			//PRINT_CUDA_MAT_SCOPE("resp" , dev_r_ , N , K , 3 );

			START_TIME(NK);

			//PRINT_CUDA_IDX("before nk SUM" , dev_r_, N * K  );


			SUM(N,K,dev_r_,dev_K_,cpy_sum,fold_sum);

			//PRINT_CUDA_MAT("nk" , dev_K_ , 1 , K  );

			/*
			//compute_Nk
			for( ITR_SIZE k = 0 ; k < K ; k++ ){
				ITR_SIZE size = N;
				cpy_test <threadNum_N , type , n_type > <<< size / threadNum_N , threadNum_N >>> (  dev_r_ , dev_N_ ,N , k );
				size /= threadNum_N;
				hipDeviceSynchronize();

				// if N = Thread Num N
				if(size == 1 ){
					cpy <<<  1 , 1 >>> ( &dev_K_[k] , dev_N_ , 1 , 0  );
					continue;
				}

				while( size > 1 ){
					dim3 block_S( threadNum_N , 1 , 1 );
					dim3 grid_S( size / block_S.x , 1 , 1 );
					if( size == threadNum_N ){
						substitute_sum_with_idx< threadNum_N , n_type , type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_K_ , size , k);
					}
					else{
						substitute_sum_with_idx< threadNum_N , n_type , n_type ><<< grid_S.x , block_S  >>> ( dev_N_ , dev_N_ , size, 0);
					}
					hipDeviceSynchronize();
					size = size / threadNum_N ;
				}
			}
			*/
#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME(NK);
#endif


			//PRINT_CUDA_MAT("nk" , dev_K_ , 1 , K  );

			type nk[K];
			hipMemcpyAsync( &nk , dev_K_ , type_width * K , hipMemcpyDeviceToHost , st_X_  );

			START_TIME(MAKE_HANDLE);
			hipblasHandle_t handle;
			hipblasCreate(&handle);
			END_TIME(MAKE_HANDLE);

			START_TIME(XK);
			// compute Xbar
			// dev_means_ <- XBar = r^T * X
			hipDeviceSynchronize();
			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_means_ , K); //float

#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME(XK);
#endif

			START_TIME(PRI);
			hipStreamSynchronize(st_X_);
			// update alpha , beta , nu

			/*
				 float nk_sum = 0;
				 printf("nk\n");
				 for(int k = 0 ; k < K ; k++ ){
				 nk_sum += nk[k];
				 printf("%.5f " , nk[k]);
				 }
				 printf("nk_sum:%.5f\n",nk_sum);
			 */

			for( int k = 0 ; k < K ; k++ ){
				weight_concentration_[k] = *weight_concentration_prior_ + nk[k];
				mean_precision_[k] = *mean_precision_prior_ + nk[k];
				degree_of_freedom_[k] = *degree_of_freedom_prior_ + nk[k];
			}
			hipMemcpyAsync( dev_mean_precision_ , mean_precision_ , size_mean_precision_ , hipMemcpyHostToDevice , st_mean_precision_ );
			hipMemcpyAsync( dev_degree_of_freedom_ , degree_of_freedom_ , size_degree_of_freedom_ , hipMemcpyHostToDevice , st_degree_of_freedom_ );
			hipDeviceSynchronize();

			END_TIME(PRI);
			START_TIME2(XK);

			divide_and_check_XBar <<< grid_K , block_K , sizeof(float) * K , st_means_ >>> ( dev_means_ , dev_K_ , dev_mean_prior_ , K , D );
			type xk[D*K];
			hipMemcpyAsync( &xk , dev_means_ , size_means_ , hipMemcpyDeviceToHost , st_means_ );

#ifdef PROFILE
			hipDeviceSynchronize();
			END_TIME2(XK);
#endif

			START_TIME(SK);
			//compute S
			pow_substitution_elements<type> <<< grid_K , block_K , 0 , st_means_ >>> (dev_KD_2_ , dev_means_ , K , D  );

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_XD_ , N,
					&_b,
					dev_KD_ , K); //float

			hipblasSgemm(
					handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
					K,D,N,
					&_a,
					dev_r_ ,N,
					dev_X_ , N,
					&_b,
					dev_precisions_ , K); //float

#if PROFILE
			hipDeviceSynchronize();
			END_TIME(SK);
#endif

			START_TIME(MEAN);
			hipStreamSynchronize(st_means_);
			// compute means on hst
			for( int k = 0 ; k < K ; k++ ){
				if( nk[k] > 0 ){
					for( int d = 0 ; d < D ; d++ ){
						means_[d*K+k]= ( ( *mean_precision_prior_ * mean_prior_[d]) + (nk[k] * xk[d*K+k] ) ) / mean_precision_[k];
#ifdef DEBUG
						CHECK_ERROR("calc mean on hst" , means_[d*K+k]);
#endif
					}

				}else{
					for( int d = 0 ; d < D ; d++ )
						means_[d*K+k] = mean_prior_[k];
				}

			}

			END_TIME(MEAN);

			START_TIME2(SK);
			hipDeviceSynchronize();
			hadamard_product_self<type> <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_precisions_ , dev_means_ , K , D );

			//dev_KD <- S = covariance
			estimate_gaussian_covariance_diag <<< grid_K , block_K , 0 , st_KD_ >>> ( dev_KD_ , dev_KD_2_ , dev_precisions_, dev_K_ , 1e-6, K , D );

#if PROFILE
			hipDeviceSynchronize();
			END_TIME2(SK);
#endif

			PRINT_CUDA_MAT("nk", dev_K_ , 1 , K );
			PRINT_CUDA_MAT("xk", dev_means_ , K , D );
			PRINT_CUDA_MAT("sk", dev_KD_ , K , D );

			/*
				 PRINT_HOST_VEC("weight_concentration_",weight_concentration_,K);
				 PRINT_HOST_VEC("mean_precision_",mean_precision_,K);
				 PRINT_CUDA_MAT("m_step mean_precision_" , dev_mean_precision_ , 1 , K );
				 PRINT_CUDA_MAT("m_step degree_of_freedom_" , dev_degree_of_freedom_ , 1 , K );
			 */

			START_TIME(PRECISION);
			//dev_precisions_ <- precision_cholesky
			estimate_precisions <<< grid_K , block_K , 0 , st_KD_>>> ( dev_K_ , dev_means_ , dev_KD_ , dev_mean_prior_ , dev_precisions_ , dev_degree_of_freedom_ , dev_covariance_prior , dev_mean_precision_ ,  dev_mean_precision_prior , K , D ) ;

			PRINT_CUDA_MAT("precision_cholesky" , dev_precisions_ , K , D );

#if PROFILE
			hipDeviceSynchronize();
			END_TIME(PRECISION);
#endif

			START_TIME(PTOE);

			//must do after beestimate_precisions
			hipMemcpyAsync( dev_means_ , means_ , size_means_ , hipMemcpyHostToDevice , st_KD_ );
			hipDeviceSynchronize();


			END_TIME(PTOE);
		}
};

int main(int argc , char *argv[]){
	ITR_SIZE N = 0, D = 0;
	TYPE *Data , *tmp_init_mean;

	unsigned short DoLearning = 1;
	unsigned short Initialize = 1;
	double old_LL = 0;

	bool InputFile = 0;
	char InputFileName[256];

	bool SaveResult = 0;
	char ClusteringResultFileName[256];

	WD = -1;
	HT = -1;
	unsigned short enable_x11 = 0;
	unsigned short AlwaysResetParameter = 0;
	float Threshold = 10;
	unsigned int   ColorAbstractionLevel = 1;
	ClusterNum = 2;

	ITR_SIZE K = 2;

	bool  TwiceScreenMode = 0;
	bool  UseCoordinateInfo = 0;

	for(argc--,argv++;argc;argc--,argv++){
		if(**argv == '-'){
			switch(*(*argv+1)){
				case 'w':
					sscanf((*argv+2), "%d", &WD);
					break;
				case 'h':
					sscanf((*argv+2), "%d", &HT);
					break;
				case 'a':
					sscanf((*argv+2), "%d", &ColorAbstractionLevel);
					if( ColorAbstractionLevel <= 0 && ColorAbstractionLevel >= 8 ){
						printf("ColorAbstractionLevel must larger than 0 and smaller than 9\n");
						exit(1);
					}
					break;
				case 'x':
					enable_x11 = 1;
					break;
				case 'c':
					sscanf((*argv+2), "%d", &K );
					break;
				case 't':
					sscanf((*argv+2), "%f", &Threshold );
					break;
				case 'r':
					AlwaysResetParameter = 1;
					break;
				case 'f':
					sscanf((*argv+3), "%s", InputFileName);
					InputFile = 1;
					break;
				case 'R':
					sscanf((*argv+3), "%s", ClusteringResultFileName );
					SaveResult = 1;
					break;
				case 's':
					TwiceScreenMode = 1;
					break;
				case 'u':
					UseCoordinateInfo = 1;
					break;
				default:
					printf("\nOptions\n");
					printf("  -w<num>      : Width of input image\n");
					printf("  -h<num>      : Height of input image\n");
					printf("  -c<num>      : Init cluster number. default : 2\n");
					printf("  -a<num>      : ColorAbstractionLevel. default : 1\n");
					printf("  -t<num>      : Threshold for reset\n");
					printf("  -x           : Enable X11 window\n");
					printf("  -r           : if likelihood fall below. re:initialize parameters by current frame\n");
					printf("  -u           : if you want to use Coordinate infomation. must $make D=5\n");
					printf("  -s           : TwiceScreenMode\n");
					printf("  -R <ClusteringResultFileName>  : if you give <ClusteringResultFileName>, you can save clustering result each data\n");
					printf("  -r <InputFileName>             : if you give <InputFileName>, you can input data by file.\n");
					exit(1);
					break;
			}
		}
	}
	N = BITMAP = WD * HT;
	printf("HT:%d\tWD:%d\tBITMAP:%d\tX_display:%d\n", HT , WD , BITMAP , enable_x11 );
	printf("Init ClusterNum:%d\n", K );

	std::chrono::system_clock::time_point start_c, end_c;
	double time = 0;

	SCRWD = 1;
	SCRHT = 1;
	if (enable_x11)
		x11_open(WD, HT, SCRWD, SCRHT); /*sh_video->disp_w, sh_video->disp_h, # rows of output_screen*/

	D = DIM;

	if( UseCoordinateInfo && DIM != 5 ){
		printf("If use coordinate infomation, please $make D=5\n");
		exit(1);
	}

	if( !UseCoordinateInfo && DIM != 3 ){
		printf("If don't use coordinate infomation, please $make D=3\n");
		exit(1);
	}

	if( DIM != 3 && DIM != 5 ){
		printf("Invalid Dimention\n");
		exit(1);
	}

	printf("DataSize:%d DataDim:%d\n" , N , D );

	Data = (float *)malloc(sizeof(float) * N * D );

	float *X;
	hipHostMalloc( &X , sizeof(float) * N * D );
#ifdef PROF
	hipProfilerStart();
#endif

	//take time
	/*
		 hipEvent_t start, stop;
		 hipEventCreate(&start);
		 hipEventCreate(&stop);
		 hipEventRecord(start);
	 */

	start_c = std::chrono::system_clock::now();

	class vbgmm< float , double , THREAD_N > vb(  N , K , D , MAX_ITER );
	unsigned int DisplayData[BITMAP];

	int fCount = 1;

	while(1){

		if (feof(stdin)) break;

		GetImageFromStdin( Data , D , TwiceScreenMode , ColorAbstractionLevel);

		if( Data == NULL ){
			printf("failed input data\n");
			return 1;
		}

		if(UseCoordinateInfo){
			for( int d = 0 ; d < D ; d++ ){
				for( int n = 0 ; n < N ; n++ ){
					if( d < 3 )
						X[ d * N + n ] = Data[ n * D + d];
					else if( d == 4 )
						X[ d * N + n ] = (n % WD) / ( (float)HT / 10 ); // Y座標入力
					else
						X[ d * N + n ] = (n / WD) / ( (float)WD / 10 ); // X座標入力
				}
			}
		}
		else{
			for( int d = 0 ; d < D ; d++ ){
				for( int n = 0 ; n < N ; n++ ){
					X[ d * N + n ] = Data[ n * D + d];
				}
			}
		}

		vb.tranport_X(X);
		if( DoLearning ){

			if( Initialize ){

				puts("start set_prior");
				vb.set_prior(X);
				puts("fin set_prior");

				Initialize = 0;

			} // if( Initialize )

			puts("start fit");
			vb.fit(X , 1e-7 );
			puts("fin fit");
			vb.print_param( );
			printf("time %lf[ms]\n", time);
		} // if( DoLearning )

		hipDeviceSynchronize();

		double LL =  vb.predict( X , DisplayData);

		printf("lower bound%.5lf\n", LL );

		if( DoLearning ){
			DoLearning = 0;
			old_LL = LL;
		}

		RGB_to_X(0, DisplayData);
		while (x11_checkevent());

		if( fabs( old_LL - LL ) > Threshold ){
			DoLearning = 1;
			if( AlwaysResetParameter ) Initialize = 1;
			printf("reset learning\n");
		}

		old_LL = LL;
	}

	end_c = std::chrono::system_clock::now();
	time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end_c - start_c).count() / 1000.0);

	// -R <ClusteringResultFileName>が存在すれば保存する

	start_c = std::chrono::system_clock::now();
	vb.free_memory_d();
	end_c = std::chrono::system_clock::now();

	time += static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end_c - start_c).count() / 1000.0);

	//hipEventRecord(stop);
	//hipEventSynchronize(stop);


	/*
		 float milliseconds = 0;
		 printf("cudaEvent time:%.5f\n", milliseconds );
		 hipEventElapsedTime(&milliseconds, start, stop);
	 */

	vb.free_memory_h();

#ifdef PROF
	hipProfilerStop();
#endif
	free(Data);

	PRINT_TIMES();

	return 0;
	}
